#include "hip/hip_runtime.h"
#define OMEGA_cuda 1.0

__host__
static inline f64 ho_K_cuda(const u32 n) {
	const f64 pi_factor = pow(OMEGA_cuda/M_PI,0.25);
	const f64 normalization_factor = pi_factor / sqrt(pow(2.0,n) * (f64)factorial_128_cuda(n));

	return normalization_factor;
}
