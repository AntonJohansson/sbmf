#include "hip/hip_runtime.h"
#define SQRT_PI_OVER_2 (1.2533141373155002512078826424055226265034933703049691583149617881)

__host__
static inline f128 hermite_integral_3_cuda(u32 i, u32 j, u32 k) {
	if ((i+j+k) % 2 != 0)
		return 0.0;

	f128 f1 = double_factorial_128_cuda( i + j - k - 1);
	f128 f2 = double_factorial_128_cuda( i - j + k - 1);
	f128 f3 = double_factorial_128_cuda(-i + j + k - 1);
	return f1*f2*f3;
}

__host__
static inline f64 hermite_integral_4_cuda(u32 i, u32 j, u32 k, u32 l) {
	u32 m_max = (i < j) ? i : j;

	f128 sum = 0.0;
	for (u32 m = 0; m <= m_max; ++m) {
		f128 integral = hermite_integral_3_cuda(i+j-2*m, k, l);
		if (integral == 0)
			continue;

		f128 b1 = n_choose_k_cuda(i, m);
		f128 b2 = n_choose_k_cuda(j, m);
		f128 m_fact = factorial_128_cuda(m);

		sum += (ho_K_cuda(i)*b1)*(ho_K_cuda(j)*b2)*(ho_K_cuda(k)*m_fact)*(ho_K_cuda(l)*pow(2.0,m))*integral;
	}

	return SQRT_PI_OVER_2*sum;
}
