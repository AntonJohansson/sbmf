#include "hip/hip_runtime.h"
__host__ __device__
static inline void map_to_triangular_index(u32 k, u32 N, u32* m, u32* n) {
	*m = k / N;
	*n = k % N;
	if (*m > *n) {
		*m = N - *m - 0;
		*n = N - *n - 1;
	}
}
/*
 * Holds all information needed to do the PT,
 * easy to pass around. Basicly params to
 * pt_rayleigh_schroedinger
 */
struct pt_settings {
	struct nlse_result* res;
	struct nlse_settings* settings;
	f64* g0;
	i64* particle_count;

	struct eigen_result_real* states;
	const u32 N; /* states to include */
	const u32 L; /* coeff count */
	nlse_operator_func* pert;
	f64* hermite_cache;
};

__host__ __device__
static inline f64 G0(struct pt_settings* pt, u32 A, u32 B) {
	return pt->g0[A*pt->res->component_count + B];
}

__host__ __device__
static inline f64 E(struct pt_settings* pt, u32 A, u32 i) {
	return pt->states[A].eigenvalues[i];
}

__host__ __device__
static inline f64* PHI(struct pt_settings* pt, u32 A, u32 i) {
	return &pt->states[A].eigenvectors[i * pt->L];
}

__host__ __device__
static inline f64 V_closed(const f64* cache, const f64* phi_a, const f64* phi_b, const f64* phi_c, const f64* phi_d, const u32 size) {
	f64 sum = 0.0;
	for (u32 a = 0; a < size; ++a) {
		for (u32 b = 0; b < size; ++b) {
			for (u32 c = 0; c < size; ++c) {
				for (u32 d = 0; d < size; ++d) {
					f64 L = phi_a[a]*phi_b[b]*phi_c[c]*phi_d[d];//*ho_K(a)*ho_K(b)*ho_K(c)*ho_K(d);
					if (fabs(L) < 1e-10)
						continue;
					f64 integral = cache[index4_cuda(a,b,c,d)];
					sum += L*integral;
				}
			}
		}
	}

	return sum;
}

/*
 * Helper functions since these will be calculated a lot
 */

__host__ __device__
static inline f64 rs_2nd_order_me(struct pt_settings* pt, u32 A, u32 B, u32 i, u32 j) {
	f64 me = 0.0;
	if (A == B) {
		f64 factor = (i == j) ? 1.0/sqrt(2.0) : 1.0;
		me = factor * G0(pt,A,A) * sqrt((f64)(pt->particle_count[A] * (pt->particle_count[A] - 1)))
			* V_closed(pt->hermite_cache, PHI(pt,A,i), PHI(pt,A,j), PHI(pt,A,0), PHI(pt,A,0), pt->L);
	} else {
		me = G0(pt,A,B) * sqrt((f64) (pt->particle_count[A] * pt->particle_count[B]))
			* V_closed(pt->hermite_cache, PHI(pt,A,i), PHI(pt,B,j), PHI(pt,A,0), PHI(pt,B,0), pt->L);
	}
	return me;
}

__host__ __device__
static inline f64 rs_2nd_order_ediff(struct pt_settings* pt, u32 A, u32 B, u32 i, u32 j) {
	return E(pt,A,0) + E(pt,B,0) - E(pt,A,i) - E(pt,B,j);
}

__global__
static void device_sum_reduction(f64* out, f64* arr, const u32 len) {
	f64 sum = 0.0;
	for (u32 i = 0; i < len; ++i) {
		sum += arr[i];
	}
	*out = sum;
}

enum pt_mode {
	MODE_RSPT = 0,
	MODE_ENPT = 1,
};

__global__
static void rspt_3_mnpq(
		enum pt_mode mode,
		f64 g,
		const u32 num_sb_states, const u32 num_mb_states, const u32 num_interactions,
		f64* pt2_cache,
		f64* hermite_cache,
		f64* coeffs,
		f64* output
		) {
	const f64 c_root_2_minus_2 = sqrt(2.0) - 2.0;
	const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

	const u32 k = blockIdx.x*blockDim.x + threadIdx.x;
	if (k >= num_interactions)
		return;
	u32 k0, k1;
	map_to_triangular_index(k, num_mb_states, &k0, &k1);

	f64 factor = 2.0;
	if (k0 == k1) {
		if (mode == MODE_RSPT)
			factor = 1.0;
		else if (mode == MODE_ENPT)
			return;
	}

	u32 m, n;
	map_to_triangular_index(k0, num_sb_states-1, &m, &n);
	m += 1; n += 1;

	u32 p, q;
	map_to_triangular_index(k1, num_sb_states-1, &p, &q);
	p += 1; q += 1;

	const f64 tmn = pt2_cache[index2_cuda(m-1,n-1)];
	const f64 tpq = pt2_cache[index2_cuda(p-1,q-1)];

	const f64 delta_mn = (m == n) ? 1.0 : 0.0;
	const f64 delta_pq = (p == q) ? 1.0 : 0.0;

	f64 v_mn_pq = g*V_closed(hermite_cache,
			&coeffs[m*num_sb_states],
			&coeffs[n*num_sb_states],
			&coeffs[p*num_sb_states],
			&coeffs[q*num_sb_states],
			num_sb_states);


	const f64 coeff = 2.0 + c_root_2_minus_2*(delta_mn + delta_pq) + c_3_minus_2_root_2*(delta_mn*delta_pq);
	output[k] = factor*coeff*tmn*tpq*v_mn_pq;
}

/*
 * Main function for Rayleigh-Schrodinger perturbation theory
 */
//
//struct pt_result rspt_1comp_cuda(struct nlse_settings settings, struct nlse_result res, u32 component, f64* g0, i64* particle_count) {
//	/* order of hamiltonians, that is include all states */
//	const u32 states_to_include = res.coeff_count;
//	const i64* N = particle_count;
//	sbmf_log_info("running 1comp RSPT cuda:\n    components: %u\n    states: %u\n", res.component_count, states_to_include);
//
//	struct eigen_result_real states;
//	states = find_eigenpairs_full_real(res.hamiltonian[component]);
//	for (u32 j = 0; j < states_to_include; ++j) {
//		f64_normalize(&states.eigenvectors[j*res.coeff_count], &states.eigenvectors[j*res.coeff_count], res.coeff_count);
//	}
//
//	f64* device_states;
//	hipMalloc(&device_states, states_to_include*states_to_include*sizeof(f64));
//	hipMemcpy(device_states, states.eigenvectors, states_to_include*states_to_include*sizeof(f64), hipMemcpyHostToDevice);
//
//	const u64 hermite_integral_count = size4_cuda(states_to_include);
//	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
//	u32 memory_marker = sbmf_stack_marker();
//	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
//	{
//		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
//		for (u32 i = 0; i < states_to_include; ++i) {
//			for (u32 j = i; j < states_to_include; ++j) {
//				for (u32 k = j; k < states_to_include; ++k) {
//					for (u32 l = k; l < states_to_include; ++l) {
//						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
//					}
//				}
//			}
//		}
//	}
//
//	f64* hermite_cache_device;
//	hipMalloc(&hermite_cache_device, hermite_cache_size);
//	hipMemcpy(hermite_cache_device, hermite_cache, hermite_cache_size, hipMemcpyHostToDevice);
//
//	struct pt_settings pt = {
//		.res = &res,
//		.settings = &settings,
//		.g0 = g0,
//		.particle_count = particle_count,
//		.states = &states,
//		.N = states_to_include,
//		.L = res.coeff_count,
//		.pert = settings.spatial_pot_perturbation,
//		.hermite_cache = hermite_cache,
//	};
//
//	/* Zeroth order PT */
//	sbmf_log_info("Starting zeroth order PT");
//	f64 E0 = 0.0;
//	{
//		E0 += N[component] * E(&pt, component, 0);
//	}
//	sbmf_log_info("\tE0: %e", E0);
//
//	/* First order PT */
//	sbmf_log_info("Starting first order PT");
//	f64 E1 = 0.0;
//	{
//		/* Handles interaction within component */
//		E1 += -0.5 * G0(&pt,component,component) * N[component] * (N[component]-1) * V_closed(hermite_cache, PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), states_to_include);
//	}
//	sbmf_log_info("\tE1: %e", E1);
//
//	const u32 pt2_cache_size = ((states_to_include-1)*(states_to_include))/2;
//	f64 pt2_cache[pt2_cache_size];
//
//	/* Assumes i in [0,states_to_include), j in [0,states_to_include) */
//#define PT2_CACHE_INDEX(i, j) \
//	((i)*states_to_include - (((i)*(i+1))/2) + j)
//
//
//	/* Second order PT */
//	sbmf_log_info("Starting second order PT");
//	f64 E2 = 0.0;
//	{
//		/*
//		 * Double substitutions (both excitations within same component),
//		 * loop over unique pairs (j,k).
//		 */
//#pragma omp parallel for reduction(+: E2)
//		for (u32 i = 1; i < states_to_include; ++i) {
//			for (u32 j = i; j < states_to_include; ++j) {
//				f64 me = rs_2nd_order_me(&pt, component,component, i,j);
//				f64 Ediff = rs_2nd_order_ediff(&pt, component,component, i,j);
//
//				pt2_cache[PT2_CACHE_INDEX(i-1, j-i)] = me/Ediff;
//
//				E2 += me*me/(Ediff);
//			}
//		}
//	}
//	sbmf_log_info("\tE2: %e", E2);
//
//	f64* device_pt2_cache;
//	hipMalloc(&device_pt2_cache, pt2_cache_size*sizeof(f64));
//	hipMemcpy(device_pt2_cache, pt2_cache, pt2_cache_size*sizeof(f64), hipMemcpyHostToDevice);
//
//	/* Third order PT */
//	sbmf_log_info("Starting third order PT");
//	f64 E3 = 0.0;
//	{
//		f64 E_00_00 = 0;
//		{
//#pragma omp parallel for reduction(+: E_00_00)
//			for (u32 m = 1; m < states_to_include; ++m) {
//				for (u32 n = m; n < states_to_include; ++n) {
//					f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1,n-m)];
//					E_00_00 += tmn*tmn;
//				}
//			}
//
//			f64 v_0000 = V_closed(hermite_cache, PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), states_to_include);
//			E_00_00 *= G0(&pt,component,component)*v_0000;
//		}
//		sbmf_log_info("\t\t00,00: %.10e", E_00_00);
//
//		f64 E_m0_n0 = 0;
//		{
//			const f64 c_root_2_minus_1 = sqrt(2.0) - 1.0;
//			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);
//
////#pragma omp parallel for reduction(+: E_m0_n0)
//			for (u32 k = 0; k < ((states_to_include-1)*states_to_include)/2; ++k) {
//				u32 m = k/(states_to_include-1);
//				u32 n = k%(states_to_include-1);
//				if (m > n) {
//					m = (states_to_include-1) - m - 0;
//					n = (states_to_include-1) - n - 1;
//				}
//				m += 1;
//				n += 1;
//
//				const f64 v_m0_n0 = G0(&pt,component,component)*V_closed(hermite_cache, PHI(&pt,component,m), PHI(&pt,component,0), PHI(&pt,component,n), PHI(&pt,component,0), states_to_include);
//
//				f64 sum = 0.0;
//				for (u32 p = 1; p < states_to_include; ++p) {
//
//					f64 tmp = 0;
//					if (p >= m)
//						tmp = pt2_cache[PT2_CACHE_INDEX(m-1, p-m)];
//					else
//						tmp = pt2_cache[PT2_CACHE_INDEX(p-1, m-p)];
//
//					f64 tnp = 0;
//					if (p >= n)
//						tnp = pt2_cache[PT2_CACHE_INDEX(n-1, p-n)];
//					else
//						tnp = pt2_cache[PT2_CACHE_INDEX(p-1, n-p)];
//
//					const f64 delta_mp = (m == p) ? 1.0 : 0.0;
//					const f64 delta_np = (n == p) ? 1.0 : 0.0;
//
//					const f64 coeff = 1 + c_root_2_minus_1*(delta_mp + delta_np) + c_3_minus_2_root_2*(delta_mp*delta_np);
//					sum += coeff * tmp * tnp;
//				}
//
//				f64 factor = (m == n) ? 1.0 : 2.0;
//
//				E_m0_n0 += factor * v_m0_n0 * sum;
//			}
//
//			E_m0_n0 *= (N[component] - 3);
//		}
//		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);
//
//		f64 E_mn_pq = 0;
//		{
//
//			const u32 num_mb_states = ((states_to_include-1)*states_to_include)/2;
//			const u32 num_interactions = (num_mb_states*(num_mb_states+1))/2;
//
//			f64* device_output;
//			hipMalloc(&device_output, num_interactions*sizeof(f64));
//
//			const u32 blocks = num_interactions/256 + 1;
//			rspt_3_mnpq<<<blocks, 256>>>(
//					MODE_RSPT,
//					g0[0],
//					states_to_include, num_mb_states, num_interactions,
//					device_pt2_cache,
//					hermite_cache_device,
//					device_states,
//					device_output
//					);
//
//			f64* res;
//			hipMalloc(&res, sizeof(f64));
//			device_sum_reduction<<<1,1>>>(res, device_output, num_interactions);
//			hipMemcpy(&E_mn_pq, res, sizeof(f64), hipMemcpyDeviceToHost);
//			hipFree(res);
//
//			hipFree(device_output);
//		}
//
//		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);
//
//		E3 = E_00_00 + E_m0_n0 + E_mn_pq;
//	}
//	sbmf_log_info("\tE3: %e", E3);
//
//	hipFree(hermite_cache_device);
//	hipFree(device_states);
//	hipFree(device_pt2_cache);
//
//	sbmf_stack_free_to_marker(memory_marker);
//
//	return (struct pt_result) {
//		.E0 = E0,
//		.E1 = E1,
//		.E2 = E2,
//		.E3 = E3,
//	};
//}
//










/*********************************************************************************************/
/***								ENPT												   ***/
/*********************************************************************************************/

struct Vp_params {
	u32 coeff_count;
	f64* i;
	f64* j;
	nlse_operator_func* pert;
};

void Vp_integrand(f64* out, f64* in, u32 len, void* data);

static inline f64 en_nhn(struct pt_settings* pt, u32 A, u32 i, u32 j) {
	f64 sum = 0;
	for (u32 k = 0; k < pt->L; ++k) {
		sum += pt->states[A].eigenvectors[i*pt->L + k] * pt->states[A].eigenvectors[j*pt->L + k] * ho_eigenval(k);

	}

	/*
	 * In the case that we're dealing with a perturbation to the
	 * basis potential, we need to compute <i|Vp|j> numerically
	 * with Vp being the pertubation
	 */

	if (pt->pert) {
		struct Vp_params p = {
			.coeff_count = pt->L,
			.i = PHI(pt, A, i),
			.j = PHI(pt, A, j),
			.pert = pt->pert,
		};

		struct quadgk_settings settings = {
			.gk = gk20,
			.abs_error_tol = 1e-15,
			.rel_error_tol = 1e-15,
			.max_iters = pt->settings->max_quadgk_iters,
			.userdata = &p,
		};

		u8 quadgk_memory[quadgk_required_memory_size(&settings)];

		struct quadgk_result res;
		quadgk_infinite_interval(Vp_integrand, &settings, quadgk_memory, &res);
		assert(res.converged);

		sum += res.integral;
	}

	return sum;
}

static inline f64 en_nhn_new(f64* phi_m, f64* phi_n, const u32 num_sb_states, nlse_operator_func* pert) {
	f64 sum = 0;
	for (u32 k = 0; k < num_sb_states; ++k) {
		sum += phi_m[k]*phi_n[k]*ho_eigenval(k);

	}

	/*
	 * In the case that we're dealing with a perturbation to the
	 * basis potential, we need to compute <i|Vp|j> numerically
	 * with Vp being the pertubation
	 */

	if (pert) {
		struct Vp_params p = {
			.coeff_count = num_sb_states,
			.i = phi_m,
			.j = phi_n,
			.pert = pert,
		};

		struct quadgk_settings settings = {
			.gk = gk20,
			.abs_error_tol = 1e-15,
			.rel_error_tol = 1e-15,
			.max_iters = 500,
			.userdata = &p,
		};

		u8 quadgk_memory[quadgk_required_memory_size(&settings)];

		struct quadgk_result res;
		quadgk_infinite_interval(Vp_integrand, &settings, quadgk_memory, &res);
		assert(res.converged);

		sum += res.integral;
	}

	return sum;
}
//
//struct pt_result enpt_1comp_cuda(struct nlse_settings settings, struct nlse_result res, u32 component, f64* g0, i64* particle_count) {
//	/* order of hamiltonians, that is include all states */
//	const u32 states_to_include = res.coeff_count;
//	const i64* N = particle_count;
//	sbmf_log_info("running 1comp ENPT:\n    components: %u\n    states: %u\n", res.component_count, states_to_include);
//	const u32 A = 0;
//
//	struct eigen_result_real states; states = find_eigenpairs_full_real(res.hamiltonian[component]); for (u32 j = 0; j < states_to_include; ++j) { f64_normalize(&states.eigenvectors[j*res.coeff_count], &states.eigenvectors[j*res.coeff_count], res.coeff_count);
//	}
//
//	f64* device_states;
//	hipMalloc(&device_states, states_to_include*states_to_include*sizeof(f64));
//	hipMemcpy(device_states, states.eigenvectors, states_to_include*states_to_include*sizeof(f64), hipMemcpyHostToDevice);
//	const u64 hermite_integral_count = size4_cuda(states_to_include);
//	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
//	u32 memory_marker = sbmf_stack_marker();
//	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
//	{
//		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
//		for (u32 i = 0; i < states_to_include; ++i) {
//			for (u32 j = i; j < states_to_include; ++j) {
//				for (u32 k = j; k < states_to_include; ++k) {
//					for (u32 l = k; l < states_to_include; ++l) {
//						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
//					}
//				}
//			}
//		}
//	}
//
//	f64* hermite_cache_device;
//	hipMalloc(&hermite_cache_device, hermite_cache_size);
//	hipMemcpy(hermite_cache_device, hermite_cache, hermite_cache_size, hipMemcpyHostToDevice);
//
//	struct pt_settings pt = {
//		.res = &res,
//		.settings = &settings,
//		.g0 = g0,
//		.particle_count = particle_count,
//		.states = &states,
//		.N = states_to_include,
//		.L = res.coeff_count,
//		.pert = settings.spatial_pot_perturbation,
//		.hermite_cache = hermite_cache,
//	};
//
//	/* Zeroth order PT */
//	sbmf_log_info("Starting zeroth order PT");
//	f64 E0 = 0.0;
//	{
//		E0 = N[component] * en_nhn(&pt, component, 0,0) + 0.5*G0(&pt,component,component)*N[component]*(N[component]-1) *
//			V_closed(hermite_cache, PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), states_to_include);
//	}
//	sbmf_log_info("\tE0: %e", E0);
//
//	const u32 pt2_cache_size = ((states_to_include-1)*(states_to_include))/2;
//	f64 pt2_cache[pt2_cache_size];
//
//	/* Assumes i in [0,states_to_include), j in [0,states_to_include) */
//#define PT2_CACHE_INDEX(i, j) \
//	((i)*states_to_include - (((i)*(i+1))/2) + j)
//
//	/* Second order PT */
//	sbmf_log_info("Starting second order PT");
//	f64 E2 = 0.0;
//	{
//		/*
//		 * Double substitutions (both excitations within same component),
//		 * loop over unique pairs (j,k).
//		 */
//#pragma omp parallel for reduction(+: E2)
//		for (u32 m = 1; m < states_to_include; ++m) {
//			for (u32 n = m; n < states_to_include; ++n) {
//				f64 me = rs_2nd_order_me(&pt, component,component, m,n);
//				const f64 v_mn_mn = V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,n), PHI(&pt,A,m), PHI(&pt,A,n), states_to_include);
//				const f64 v_m0_m0 = V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,0), PHI(&pt,A,m), PHI(&pt,A,0), states_to_include);
//				const f64 v_n0_n0 = (m == n) ? v_m0_m0 : V_closed(hermite_cache, PHI(&pt,A,n), PHI(&pt,A,0), PHI(&pt,A,n), PHI(&pt,A,0), states_to_include);
//				const f64 v_00_00 = V_closed(hermite_cache, PHI(&pt,A,0), PHI(&pt,A,0), PHI(&pt,A,0), PHI(&pt,A,0), states_to_include);
//
//				const f64 dmn = (m == n) ? 1.0 : 0.0;
//				f64 Ediff =
//					2.0*en_nhn(&pt,A,0,0) - en_nhn(&pt,A,m,m) - en_nhn(&pt,A,n,n)
//					- G0(&pt,A,A)*((2.0-dmn)*v_mn_mn + 2.0*(N[A]-2.0)*(v_m0_m0 + v_n0_n0) - (2.0*N[A]-3.0)*v_00_00);
//
//				pt2_cache[PT2_CACHE_INDEX(m-1, n-m)] = me/Ediff;
//
//				E2 += me*me/(Ediff);
//			}
//		}
//	}
//	sbmf_log_info("\tE2: %e", E2);
//
//	f64* device_pt2_cache;
//	hipMalloc(&device_pt2_cache, pt2_cache_size*sizeof(f64));
//	hipMemcpy(device_pt2_cache, pt2_cache, pt2_cache_size*sizeof(f64), hipMemcpyHostToDevice);
//
//	/* Third order PT */
//	sbmf_log_info("Starting third order PT");
//	f64 E3 = 0.0;
//	{
//		f64 E_m0_n0 = 0;
//		{
//			const f64 c_root_2_minus_1 = sqrt(2.0) - 1.0;
//			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);
//
//#pragma omp parallel for reduction(+: E_m0_n0)
//			for (u32 k = 0; k < ((states_to_include-1)*states_to_include)/2; ++k) {
//				u32 m = k/(states_to_include-1);
//				u32 n = k%(states_to_include-1);
//
//				if (m > n) {
//					m = (states_to_include-1) - m - 0;
//					n = (states_to_include-1) - n - 1;
//				}
//
//				if (m == n)
//					continue;
//
//				m += 1;
//				n += 1;
//
//				const f64 v_m0_n0 = G0(&pt,component,component)*V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,0), PHI(&pt,A,n), PHI(&pt,A,0), states_to_include);
//
//				f64 sum = 0.0;
//				for (u32 p = 1; p < states_to_include; ++p) {
//
//					f64 tmp = 0;
//					if (p >= m)
//						tmp = pt2_cache[PT2_CACHE_INDEX(m-1, p-m)];
//					else
//						tmp = pt2_cache[PT2_CACHE_INDEX(p-1, m-p)];
//
//					f64 tnp = 0;
//					if (p >= n)
//						tnp = pt2_cache[PT2_CACHE_INDEX(n-1, p-n)];
//					else
//						tnp = pt2_cache[PT2_CACHE_INDEX(p-1, n-p)];
//
//					const f64 delta_mp = (m == p) ? 1.0 : 0.0;
//					const f64 delta_np = (n == p) ? 1.0 : 0.0;
//
//					const f64 coeff = 1 + c_root_2_minus_1*(delta_mp + delta_np) + c_3_minus_2_root_2*(delta_mp*delta_np);
//					sum += coeff * tmp * tnp;
//				}
//
//				E_m0_n0 += 2.0 * v_m0_n0 * sum;
//			}
//
//			E_m0_n0 *= (N[component] - 3);
//		}
//		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);
//
//		f64 E_mn_pq = 0;
//		{
//
//			const u32 num_mb_states = ((states_to_include-1)*states_to_include)/2;
//			const u32 num_interactions = (num_mb_states*(num_mb_states+1))/2;
//
//			f64* device_output;
//			hipMalloc(&device_output, num_interactions*sizeof(f64));
//
//			const u32 blocks = num_interactions/256 + 1;
//			rspt_3_mnpq<<<blocks, 256>>>(
//					MODE_ENPT,
//					g0[0],
//					states_to_include, num_mb_states, num_interactions,
//					device_pt2_cache,
//					hermite_cache_device,
//					device_states,
//					device_output
//					);
//
//			f64* res;
//			hipMalloc(&res, sizeof(f64));
//			device_sum_reduction<<<1,1>>>(res, device_output, num_interactions);
//			hipMemcpy(&E_mn_pq, res, sizeof(f64), hipMemcpyDeviceToHost);
//			hipFree(res);
//
//			hipFree(device_output);
//		}
//		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);
//
//		E3 = E_m0_n0 + E_mn_pq;
//	}
//	sbmf_log_info("\tE3: %e", E3);
//
//	hipFree(hermite_cache_device);
//	hipFree(device_states);
//	hipFree(device_pt2_cache);
//
//	sbmf_stack_free_to_marker(memory_marker);
//
//	return (struct pt_result) {
//		.E0 = E0,
//		.E1 = 0,
//		.E2 = E2,
//		.E3 = E3,
//	};
//}
//
static struct pt_result perturbation_theory_1comp(enum pt_mode mode, f64 g, i64 N, const f64* hermite_cache, const u32 hermite_cache_size, const struct eigen_result_real* states, const f64 groundstate_energy, const f64* double_subst_energy_diffs, const u32 num_sb_states) {
	f64* device_states;
	hipMalloc(&device_states, num_sb_states*num_sb_states*sizeof(f64));
	hipMemcpy(device_states, states->eigenvectors, num_sb_states*num_sb_states*sizeof(f64), hipMemcpyHostToDevice);

	f64* hermite_cache_device;
	hipMalloc(&hermite_cache_device, hermite_cache_size);
	hipMemcpy(hermite_cache_device, hermite_cache, hermite_cache_size, hipMemcpyHostToDevice);

	/* Zeroth order PT */
	sbmf_log_info("Starting zeroth order PT");
	f64 E0 = groundstate_energy;
	sbmf_log_info("\tE0: %e", E0);

	/* This particular integral shows up in zeroth and third order rspt */
	const f64 v_00_00 = V_closed(hermite_cache,
			&states->eigenvectors[0],
			&states->eigenvectors[0],
			&states->eigenvectors[0],
			&states->eigenvectors[0],
			num_sb_states);

	/* First order PT */
	f64 E1 = 0.0;
	if (mode == MODE_RSPT) {
		sbmf_log_info("Starting first order PT");
		E1 = -0.5*g*N*(N-1)*v_00_00;
		sbmf_log_info("\tE1: %e", E1);
	}

	const u32 pt2_cache_size = size2_cuda(num_sb_states-1);
	f64 pt2_cache[pt2_cache_size];

	/* Second order PT */
	sbmf_log_info("Starting second order PT");
	f64 E2 = 0.0;
	{
		/*
		 * Double substitutions (both excitations within same component),
		 * loop over unique pairs (j,k).
		 */
#pragma omp parallel for reduction(+: E2)
		for (u32 m = 1; m < num_sb_states; ++m) {
			for (u32 n = m; n < num_sb_states; ++n) {
				const f64 v_mn_00 = V_closed(hermite_cache,
						&states->eigenvectors[m*num_sb_states],
						&states->eigenvectors[n*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						num_sb_states);
				const f64 factor = (m == n) ? 1.0/sqrt(2.0) : 1.0;
				const f64 me = factor*g*sqrt(N*(N-1))*v_mn_00;

				const f64 Ediff = double_subst_energy_diffs[index2_cuda(m-1,n-1)];

				pt2_cache[index2_cuda(m-1,n-1)] = me/Ediff;

				E2 += me*me/(Ediff);
			}
		}
	}
	sbmf_log_info("\tE2: %e", E2);

	f64* device_pt2_cache;
	hipMalloc(&device_pt2_cache, pt2_cache_size*sizeof(f64));
	hipMemcpy(device_pt2_cache, pt2_cache, pt2_cache_size*sizeof(f64), hipMemcpyHostToDevice);

	/* Third order PT */
	sbmf_log_info("Starting third order PT");
	f64 E3 = 0.0;
	{
		f64 E_00_00 = 0;
		if (mode == MODE_RSPT) {
#pragma omp parallel for reduction(+: E_00_00)
			for (u32 m = 1; m < num_sb_states; ++m) {
				for (u32 n = m; n < num_sb_states; ++n) {
					const f64 tmn = pt2_cache[index2_cuda(m-1,n-1)];
					E_00_00 += tmn*tmn;
				}
			}

			E_00_00 *= g*v_00_00;
			sbmf_log_info("\t\t00,00: %.10e", E_00_00);
		}

		/* Number of many-body states, excludes 0,0 */
		const u32 num_mb_states = size2_cuda(num_sb_states-1);

		f64 E_m0_n0 = 0;
		{
			const f64 c_root_2_minus_1 = sqrt(2.0) - 1.0;
			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

#pragma omp parallel for reduction(+: E_m0_n0)
			for (u32 k = 0; k < num_mb_states; ++k) {
				u32 m, n;
				map_to_triangular_index(k, num_sb_states-1, &m, &n);
				if (mode == MODE_ENPT && m == n)
					continue;

				m += 1;
				n += 1;

				const f64 v_m0_n0 = g*V_closed(hermite_cache,
						&states->eigenvectors[m*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						&states->eigenvectors[n*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						num_sb_states);

				f64 sum = 0.0;
				for (u32 p = 1; p < num_sb_states; ++p) {

					const f64 tmp = pt2_cache[index2_cuda(m-1,p-1)];
					const f64 tnp = pt2_cache[index2_cuda(n-1,p-1)];

					const f64 delta_mp = (m == p) ? 1.0 : 0.0;
					const f64 delta_np = (n == p) ? 1.0 : 0.0;

					const f64 coeff = 1 + c_root_2_minus_1*(delta_mp + delta_np) + c_3_minus_2_root_2*(delta_mp*delta_np);
					sum += coeff * tmp * tnp;
				}

				f64 factor = (m == n) ? 1.0 : 2.0;

				E_m0_n0 += factor * v_m0_n0 * sum;
			}

			E_m0_n0 *= (N - 3);
		}
		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);

		f64 E_mn_pq = 0;
		{
			const u32 num_interactions = (num_mb_states*(num_mb_states+1))/2;

			f64* device_output;
			hipMalloc(&device_output, num_interactions*sizeof(f64));

			const u32 blocks = num_interactions/256 + 1;
			rspt_3_mnpq<<<blocks, 256>>>(
					mode,
					g,
					num_sb_states, num_mb_states, num_interactions,
					device_pt2_cache,
					hermite_cache_device,
					device_states,
					device_output
					);

			f64* res;
			hipMalloc(&res, sizeof(f64));
			device_sum_reduction<<<1,1>>>(res, device_output, num_interactions);
			hipMemcpy(&E_mn_pq, res, sizeof(f64), hipMemcpyDeviceToHost);
			hipFree(res);

			hipFree(device_output);
		}

		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);

		E3 = E_00_00 + E_m0_n0 + E_mn_pq;
	}
	sbmf_log_info("\tE3: %e", E3);

	hipFree(hermite_cache_device);
	hipFree(device_states);
	hipFree(device_pt2_cache);

	return (struct pt_result) {
		.E0 = E0,
		.E1 = E1,
		.E2 = E2,
		.E3 = E3,
	};
}

static struct pt_result perturbation_theory_2comp(enum pt_mode mode, f64 gAA, f64 gAB, i64 NA, i64 NB, const f64* hermite_cache, const u32 hermite_cache_size, const struct eigen_result_real* statesA, const struct eigen_result_real* statesB,
		const f64 groundstate_energy,
		const f64* double_subst_energy_diffs_AA,
		const f64* double_subst_energy_diffs_BB,
		const f64* double_subst_energy_diffs_AB,
		const u32 num_sb_states) {

	struct pt_result res_A = perturbation_theory_1comp(mode, gAA, NA, hermite_cache, hermite_cache_size, statesA, groundstate_energy, double_subst_energy_diffs_AA, num_sb_states);
	struct pt_result res_B = perturbation_theory_1comp(mode, gAA, NB, hermite_cache, hermite_cache_size, statesB, groundstate_energy, double_subst_energy_diffs_BB, num_sb_states);

	sbmf_log_info("Starting zeroth order PT");
	//f64 E0 = res_A.E0 + res_B.E0;
	f64 E0 = groundstate_energy;
	sbmf_log_info("\tE0: %e", E0);

	f64 E1 = 0.0;
	if (mode == MODE_RSPT) {
		sbmf_log_info("Starting first order PT");
		E1 = res_A.E1 + res_B.E1;
		E1 += -gAB*NA*NB*V_closed(hermite_cache,
				&statesA->eigenvectors[0],
				&statesB->eigenvectors[0],
				&statesA->eigenvectors[0],
				&statesB->eigenvectors[0],
				num_sb_states);
		sbmf_log_info("\tE1: %e", E1);
	}

	const u32 pt2_cache_size = (num_sb_states-1)*(num_sb_states-1);
	f64 pt2_cache[pt2_cache_size];

	/* Assumes i in [0,states_to_include), j in [0,states_to_include) */
#define PT2_CACHE_INDEX(i, j) \
	(i)*(num_sb_states-1) + (j)

	/* Second order PT */
	sbmf_log_info("Starting second order PT");
	f64 E2 = 0.0;
	{
		E2 = res_A.E2 + res_B.E2;
		/*
		 * Double substitutions (separate components).
		 * A,B refers to components.
		 */
#pragma omp parallel for reduction(+: E2)
		for (u32 m = 1; m < num_sb_states; ++m) {
			for (u32 n = 1; n < num_sb_states; ++n) {
				const f64 me = gAB*sqrt(NA*NB)
					* V_closed(hermite_cache,
							&statesA->eigenvectors[m*num_sb_states],
							&statesB->eigenvectors[n*num_sb_states],
							&statesA->eigenvectors[0*num_sb_states],
							&statesB->eigenvectors[0*num_sb_states],
							num_sb_states);

				const f64 Ediff = double_subst_energy_diffs_AB[(m-1)*(num_sb_states-1) + (n-1)];

				pt2_cache[PT2_CACHE_INDEX(m-1,n-1)] = me/Ediff;

				E2 += me*me/(Ediff);
			}
		}
	}
	sbmf_log_info("\tE2: %e", E2);

	sbmf_log_info("Starting third order PT");
	f64 E3 = 0.0;
	{
		E3 = res_A.E3 + res_B.E3;

		f64 E_00_00 = 0.0;
		if (mode == MODE_RSPT) {

			f64 sum = 0;
#pragma omp parallel for reduction(+: sum)
			for (u32 m = 1; m < num_sb_states; ++m) {
				for (u32 n = 1; n < num_sb_states; ++n) {
					const f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1, n-1)];
					sum += tmn*tmn;
				}
			}

			const f64 v_00_00 = V_closed(hermite_cache,
					&statesA->eigenvectors[0],
					&statesB->eigenvectors[0],
					&statesA->eigenvectors[0],
					&statesB->eigenvectors[0],
					num_sb_states);
			E_00_00 += gAB * v_00_00 * sum;
		}
		sbmf_log_info("\t\t00,00: %.10e", E_00_00);

		f64 E_m0_n0 = 0.0;
		{
			{
#pragma omp parallel for reduction(+: E_m0_n0)
				for (u32 m = 1; m < num_sb_states; ++m) {
					for (u32 n = 1; n < num_sb_states; ++n) {

						f64 sumA = 0;
						f64 sumB = 0;
						for (u32 p = 1; p < num_sb_states; ++p) {
							const f64 tmp = pt2_cache[PT2_CACHE_INDEX(m-1, p-1)];
							const f64 tnp = pt2_cache[PT2_CACHE_INDEX(n-1, p-1)];
							sumA += tmp*tnp;

							const f64 tpm = pt2_cache[PT2_CACHE_INDEX(p-1, m-1)];
							const f64 tpn = pt2_cache[PT2_CACHE_INDEX(p-1, n-1)];
							sumB += tpm*tpn;
						}

						const f64 v_AA_m0_n0 = gAA*(NA-1)*V_closed(hermite_cache,
								&statesA->eigenvectors[m*num_sb_states],
								&statesA->eigenvectors[0*num_sb_states],
								&statesA->eigenvectors[n*num_sb_states],
								&statesA->eigenvectors[0*num_sb_states],
								num_sb_states);
						const f64 v_BB_m0_n0 = gAA*(NB-1)*V_closed(hermite_cache,
								&statesB->eigenvectors[m*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								&statesB->eigenvectors[n*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								num_sb_states);
						const f64 v_AB_m0_n0 = - gAB*V_closed(hermite_cache,
								&statesA->eigenvectors[m*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								&statesA->eigenvectors[n*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								num_sb_states);
						const f64 v_AB_0m_0n = - gAB*V_closed(hermite_cache,
								&statesA->eigenvectors[0*num_sb_states],
								&statesB->eigenvectors[m*num_sb_states],
								&statesA->eigenvectors[0*num_sb_states],
								&statesB->eigenvectors[n*num_sb_states],
								num_sb_states);
						E_m0_n0 += (v_AA_m0_n0 + v_AB_m0_n0) * sumA;
						E_m0_n0 += (v_BB_m0_n0 + v_AB_0m_0n) * sumB;
					}
				}
			}
		}
		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);

		f64 E_mn_pq = 0;
		{
#pragma omp parallel for reduction(+: E_mn_pq)
			for (u32 m = 1; m < num_sb_states; ++m) {
				for (u32 n = 1; n < num_sb_states; ++n) {
					for (u32 p = 1; p < num_sb_states; ++p) {
						for (u32 q = 1; q < num_sb_states; ++q) {
							const f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1, n-1)];
							const f64 tpq = pt2_cache[PT2_CACHE_INDEX(p-1, q-1)];
							const f64 v_mn_pq = gAB * V_closed(hermite_cache,
									&statesA->eigenvectors[m*num_sb_states],
									&statesB->eigenvectors[n*num_sb_states],
									&statesA->eigenvectors[p*num_sb_states],
									&statesB->eigenvectors[q*num_sb_states],
									num_sb_states);
							E_mn_pq += v_mn_pq*tmn*tpq;
						}
					}
				}
			}
		}
		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);

		E3 += E_00_00 + E_m0_n0 + E_mn_pq;
	}
	sbmf_log_info("\tE3: %e", E3);

	return (struct pt_result) {
		.E0 = E0,
		.E1 = E1,
		.E2 = E2,
		.E3 = E3,
	};
}

/**************************************************************************************************************************************************/

struct pt_result rspt_1comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 component, f64 g, i64 N) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real states;
	states = find_eigenpairs_full_real(res.hamiltonian[component]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&states.eigenvectors[j*num_sb_states], &states.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
					}
				}
			}
		}
	}


	f64 groundstate_energy = N*states.eigenvalues[0];
	/* Energies of double substitution states including the zero states */
	f64 double_subst_energy_diffs[size2_cuda(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			double_subst_energy_diffs[index2_cuda(m-1,n-1)] = 2*states.eigenvalues[0] - states.eigenvalues[m] - states.eigenvalues[n];
		}
	}

	struct pt_result ptres = perturbation_theory_1comp(MODE_RSPT, g, N, hermite_cache, hermite_cache_size, &states, groundstate_energy, double_subst_energy_diffs, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}

struct pt_result enpt_1comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 component, f64 g, i64 N) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real states;
	states = find_eigenpairs_full_real(res.hamiltonian[component]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&states.eigenvectors[j*num_sb_states], &states.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
					}
				}
			}
		}
	}

	/* Energies of double substitution states including the zero states */
	f64 double_subst_energy_diffs[size2_cuda(num_sb_states-1)];

	const f64 v_00_00 = V_closed(hermite_cache,
			&states.eigenvectors[0],
			&states.eigenvectors[0],
			&states.eigenvectors[0],
			&states.eigenvectors[0],
			num_sb_states);
	f64 groundstate_energy = N*en_nhn_new(&states.eigenvectors[0*num_sb_states], &states.eigenvectors[0*num_sb_states], num_sb_states, settings->spatial_pot_perturbation) + 0.5*g*N*(N-1)*v_00_00;

	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			const f64 v_mn_mn = V_closed(hermite_cache,
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[n*num_sb_states],
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_m0_m0 = V_closed(hermite_cache,
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_n0_n0 = (m == n) ? v_m0_m0 : V_closed(hermite_cache,
					&states.eigenvectors[n*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					&states.eigenvectors[n*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					num_sb_states);

			const f64 dmn = (m == n) ? 1.0 : 0.0;
			f64 energy =
				2*en_nhn_new(&states.eigenvectors[0*num_sb_states], &states.eigenvectors[0*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				-en_nhn_new(&states.eigenvectors[m*num_sb_states], &states.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				-en_nhn_new(&states.eigenvectors[n*num_sb_states], &states.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				- g*((2.0-dmn)*v_mn_mn + 2.0*(N-2)*(v_m0_m0+v_n0_n0) - (2*N-3)*v_00_00);

			double_subst_energy_diffs[index2_cuda(m-1,n-1)] = energy;
		}
	}

	struct pt_result ptres = perturbation_theory_1comp(MODE_ENPT, g, N, hermite_cache, hermite_cache_size, &states, groundstate_energy, double_subst_energy_diffs, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}

struct pt_result rspt_2comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 compA, u32 compB, f64 gAA, f64 gAB, i64 NA, i64 NB) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real statesA, statesB;
	statesA = find_eigenpairs_full_real(res.hamiltonian[compA]);
	statesB = find_eigenpairs_full_real(res.hamiltonian[compB]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&statesA.eigenvectors[j*num_sb_states], &statesA.eigenvectors[j*num_sb_states], num_sb_states);
		f64_normalize(&statesB.eigenvectors[j*num_sb_states], &statesB.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
					}
				}
			}
		}
	}

	const f64 groundstate_energy = NA*statesA.eigenvalues[0] + NB*statesB.eigenvalues[0];

	/* Energies of double substitution states including the zero states */
	f64 double_subst_energies_AA[size2_cuda(num_sb_states-1)];
	f64 double_subst_energies_BB[size2_cuda(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			double_subst_energies_AA[index2_cuda(m-1,n-1)] = 2*statesA.eigenvalues[0] - statesA.eigenvalues[m] - statesA.eigenvalues[n];
			double_subst_energies_BB[index2_cuda(m-1,n-1)] = 2*statesB.eigenvalues[0] - statesB.eigenvalues[m] - statesB.eigenvalues[n];
		}
	}

	f64 double_subst_energies_AB[(num_sb_states-1)*(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = 1; n < num_sb_states; ++n) {
			double_subst_energies_AB[(m-1)*(num_sb_states-1) + (n-1)] = statesA.eigenvalues[0] + statesB.eigenvalues[0] - statesA.eigenvalues[m] - statesB.eigenvalues[n];
		}
	}

	struct pt_result ptres = perturbation_theory_2comp(MODE_RSPT, gAA, gAB, NA, NB, hermite_cache, hermite_cache_size, &statesA, &statesB, groundstate_energy, double_subst_energies_AA, double_subst_energies_BB, double_subst_energies_AB, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}

struct pt_result enpt_2comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 compA, u32 compB, f64 gAA, f64 gAB, i64 NA, i64 NB) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real statesA, statesB;
	statesA = find_eigenpairs_full_real(res.hamiltonian[compA]);
	statesB = find_eigenpairs_full_real(res.hamiltonian[compB]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&statesA.eigenvectors[j*num_sb_states], &statesA.eigenvectors[j*num_sb_states], num_sb_states);
		f64_normalize(&statesB.eigenvectors[j*num_sb_states], &statesB.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
					}
				}
			}
		}
	}

	const f64 v_AA_00_00 = V_closed(hermite_cache,
			&statesA.eigenvectors[0],
			&statesA.eigenvectors[0],
			&statesA.eigenvectors[0],
			&statesA.eigenvectors[0],
			num_sb_states);
	const f64 v_BB_00_00 = V_closed(hermite_cache,
			&statesB.eigenvectors[0],
			&statesB.eigenvectors[0],
			&statesB.eigenvectors[0],
			&statesB.eigenvectors[0],
			num_sb_states);
	const f64 v_AB_00_00 = V_closed(hermite_cache,
			&statesA.eigenvectors[0],
			&statesB.eigenvectors[0],
			&statesA.eigenvectors[0],
			&statesB.eigenvectors[0],
			num_sb_states);
	const f64 groundstate_energy =
		NA*en_nhn_new(&statesA.eigenvectors[0], &statesA.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation) + 0.5*gAA*NA*(NA-1)*v_AA_00_00
		+ NB*en_nhn_new(&statesB.eigenvectors[0], &statesB.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation) + 0.5*gAA*NB*(NB-1)*v_BB_00_00
		+ gAB*NA*NB*v_AB_00_00;

	/* Energies of double substitution states including the zero states */
	f64 double_subst_energy_diffs_AA[size2_cuda(num_sb_states-1)];
	f64 double_subst_energy_diffs_BB[size2_cuda(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			f64 delta_mn = (m == n) ? 1.0 : 0.0;
			const f64 v_AA_mn_mn = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_AA_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AA_n0_n0 = (m == n) ? v_AA_m0_m0 : V_closed(hermite_cache,
					&statesA.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_n0_n0 = (m == n) ? v_AB_m0_m0 : V_closed(hermite_cache,
					&statesA.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			double_subst_energy_diffs_AA[index2_cuda(m-1,n-1)] =
				  2*en_nhn_new(&statesA.eigenvectors[0], &statesA.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesA.eigenvectors[m*num_sb_states], &statesA.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesA.eigenvectors[n*num_sb_states], &statesA.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*((2.0-delta_mn)*v_AA_mn_mn + 2.0*(NA-2)*(v_AA_m0_m0 + v_AA_n0_n0) - (2.0*NA-3.0)*v_AA_00_00)
				  - gAB*NB*(v_AB_m0_m0 + v_AB_n0_n0 - 2.0*v_AB_00_00);


			const f64 v_BB_mn_mn = V_closed(hermite_cache,
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_BB_m0_m0 = V_closed(hermite_cache,
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_BB_n0_n0 = (m == n) ? v_AA_m0_m0 : V_closed(hermite_cache,
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_0m_0m = V_closed(hermite_cache,
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					num_sb_states);
			const f64 v_AB_0n_0n = (m == n) ? v_AB_m0_m0 : V_closed(hermite_cache,
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			double_subst_energy_diffs_BB[index2_cuda(m-1,n-1)] =
				  2*en_nhn_new(&statesB.eigenvectors[0], &statesB.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesB.eigenvectors[m*num_sb_states], &statesB.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesB.eigenvectors[n*num_sb_states], &statesB.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*((2.0-delta_mn)*v_BB_mn_mn + 2.0*(NB-2)*(v_BB_m0_m0 + v_BB_n0_n0) - (2.0*NB-3.0)*v_BB_00_00)
				  - gAB*NB*(v_AB_0m_0m + v_AB_0n_0n - 2.0*v_AB_00_00);
		}
	}

	f64 double_subst_energy_diffs_AB[(num_sb_states-1)*(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = 1; n < num_sb_states; ++n) {
			const f64 v_AA_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_BB_n0_n0 = V_closed(hermite_cache,
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_mn_mn = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_AB_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_0n_0n = V_closed(hermite_cache,
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			double_subst_energy_diffs_AB[(m-1)*(num_sb_states-1) + (n-1)] =
				    en_nhn_new(&statesA.eigenvectors[0], &statesA.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesA.eigenvectors[m*num_sb_states], &statesA.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*(2.0*(NA-1)*v_AA_m0_m0 - (NA-1)*v_AA_00_00)
				  + en_nhn_new(&statesB.eigenvectors[0], &statesB.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesB.eigenvectors[n*num_sb_states], &statesB.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*(2.0*(NB-1)*v_BB_n0_n0 - (NB-1)*v_BB_00_00)
				  - gAB*(v_AB_mn_mn + (NB-1)*v_AB_m0_m0 + (NA-1)*v_AB_0n_0n - (NA + NB - 1)*v_AB_00_00);
		}
	}

	struct pt_result ptres = perturbation_theory_2comp(MODE_ENPT, gAA, gAB, NA, NB, hermite_cache, hermite_cache_size, &statesA, &statesB, groundstate_energy, double_subst_energy_diffs_AA, double_subst_energy_diffs_BB, double_subst_energy_diffs_AB, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}
