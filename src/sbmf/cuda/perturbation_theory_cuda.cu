#include "hip/hip_runtime.h"
__host__ __device__
static inline void map_to_triangular_index(u32 k, u32 N, u32* m, u32* n) {
	*m = k / N;
	*n = k % N;
	if (*m > *n) {
		*m = N - *m - 0;
		*n = N - *n - 1;
	}
}
/*
 * Holds all information needed to do the PT,
 * easy to pass around. Basicly params to
 * pt_rayleigh_schroedinger
 */
struct pt_settings {
	struct nlse_result* res;
	struct nlse_settings* settings;
	f64* g0;
	i64* particle_count;

	struct eigen_result_real* states;
	const u32 N; /* states to include */
	const u32 L; /* coeff count */
	nlse_operator_func* pert;
	f64* hermite_cache;
};

__host__ __device__
static inline f64 G0(struct pt_settings* pt, u32 A, u32 B) {
	return pt->g0[A*pt->res->component_count + B];
}

__host__ __device__
static inline f64 E(struct pt_settings* pt, u32 A, u32 i) {
	return pt->states[A].eigenvalues[i];
}

__host__ __device__
static inline f64* PHI(struct pt_settings* pt, u32 A, u32 i) {
	return &pt->states[A].eigenvectors[i * pt->L];
}

__host__ __device__
static inline f64 V_closed(f64* cache, f64* phi_a, f64* phi_b, f64* phi_c, f64* phi_d, const u32 size) {
	f64 sum = 0.0;
	for (u32 a = 0; a < size; ++a) {
		for (u32 b = 0; b < size; ++b) {
			for (u32 c = 0; c < size; ++c) {
				for (u32 d = 0; d < size; ++d) {
					f64 L = phi_a[a]*phi_b[b]*phi_c[c]*phi_d[d];//*ho_K(a)*ho_K(b)*ho_K(c)*ho_K(d);
					if (fabs(L) < 1e-10)
						continue;
					f64 integral = cache[index4_cuda(a,b,c,d)];
					sum += L*integral;
				}
			}
		}
	}

	return sum;
}

/*
 * Helper functions since these will be calculated a lot
 */

__host__ __device__
static inline f64 rs_2nd_order_me(struct pt_settings* pt, u32 A, u32 B, u32 i, u32 j) {
	f64 me = 0.0;
	if (A == B) {
		f64 factor = (i == j) ? 1.0/sqrt(2.0) : 1.0;
		me = factor * G0(pt,A,A) * sqrt((f64)(pt->particle_count[A] * (pt->particle_count[A] - 1)))
			* V_closed(pt->hermite_cache, PHI(pt,A,i), PHI(pt,A,j), PHI(pt,A,0), PHI(pt,A,0), pt->L);
	} else {
		me = G0(pt,A,B) * sqrt((f64) (pt->particle_count[A] * pt->particle_count[B]))
			* V_closed(pt->hermite_cache, PHI(pt,A,i), PHI(pt,B,j), PHI(pt,A,0), PHI(pt,B,0), pt->L);
	}
	return me;
}

__host__ __device__
static inline f64 rs_2nd_order_ediff(struct pt_settings* pt, u32 A, u32 B, u32 i, u32 j) {
	return E(pt,A,0) + E(pt,B,0) - E(pt,A,i) - E(pt,B,j);
}

__global__
static void device_sum_reduction(f64* out, f64* arr, const u32 len) {
	f64 sum = 0.0;
	for (u32 i = 0; i < len; ++i) {
		sum += arr[i];
	}
	*out = sum;
}

__global__
static void rspt_3_mnpq(
		f64* g0,
		const u32 num_sb_states, const u32 num_mb_states, const u32 num_interactions,
		f64* pt2_cache,
		f64* hermite_cache,
		f64* coeffs,
		f64* output
		) {
	const f64 c_root_2_minus_2 = sqrt(2.0) - 2.0;
	const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

	const u32 k = blockIdx.x*blockDim.x + threadIdx.x;
	if (k >= num_interactions)
		return;
	u32 k0, k1;
	map_to_triangular_index(k, num_mb_states, &k0, &k1);

	u32 m, n;
	map_to_triangular_index(k0, num_sb_states-1, &m, &n);
	m += 1; n += 1;

	u32 p, q;
	map_to_triangular_index(k1, num_sb_states-1, &p, &q);
	p += 1; q += 1;

	f64 factor = 2.0;
	if (k0 == k1)
		factor = 1.0;

#define PT2_CACHE_INDEX(i, j) \
	((i)*num_sb_states - (((i)*(i+1))/2) + j)

	f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1,n-m)];
	f64 tpq = pt2_cache[PT2_CACHE_INDEX(p-1,q-p)];
#undef PT2_CACHE_INDEX

	const f64 delta_mn = (m == n) ? 1.0 : 0.0;
	const f64 delta_pq = (p == q) ? 1.0 : 0.0;

	f64 v_mn_pq = (*g0)*V_closed(hermite_cache,
			&coeffs[m*num_sb_states],
			&coeffs[n*num_sb_states],
			&coeffs[p*num_sb_states],
			&coeffs[q*num_sb_states],
			num_sb_states);


	const f64 coeff = 2.0 + c_root_2_minus_2*(delta_mn + delta_pq) + c_3_minus_2_root_2*(delta_mn*delta_pq);
	output[k] = factor*coeff*tmn*tpq*v_mn_pq;
}

/*
 * Main function for Rayleigh-Schrodinger perturbation theory
 */

struct pt_result rspt_1comp_cuda(struct nlse_settings settings, struct nlse_result res, u32 component, f64* g0, i64* particle_count) {
	/* order of hamiltonians, that is include all states */
	const u32 states_to_include = res.coeff_count;
	const i64* N = particle_count;
	sbmf_log_info("running 1comp RSPT cuda:\n    components: %u\n    states: %u\n", res.component_count, states_to_include);

	struct eigen_result_real states;
	states = find_eigenpairs_full_real(res.hamiltonian[component]);
	for (u32 j = 0; j < states_to_include; ++j) {
		f64_normalize(&states.eigenvectors[j*res.coeff_count], &states.eigenvectors[j*res.coeff_count], res.coeff_count);
	}

	f64* device_states;
	hipMalloc(&device_states, states_to_include*states_to_include*sizeof(f64));
	hipMemcpy(device_states, states.eigenvectors, states_to_include*states_to_include*sizeof(f64), hipMemcpyHostToDevice);

	const u64 hermite_integral_count = size4_cuda(states_to_include);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < states_to_include; ++i) {
			for (u32 j = i; j < states_to_include; ++j) {
				for (u32 k = j; k < states_to_include; ++k) {
					for (u32 l = k; l < states_to_include; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
					}
				}
			}
		}
	}

	f64* hermite_cache_device;
	hipMalloc(&hermite_cache_device, hermite_cache_size);
	hipMemcpy(hermite_cache_device, hermite_cache, hermite_cache_size, hipMemcpyHostToDevice);

	struct pt_settings pt = {
		.res = &res,
		.settings = &settings,
		.g0 = g0,
		.particle_count = particle_count,
		.states = &states,
		.N = states_to_include,
		.L = res.coeff_count,
		.pert = settings.spatial_pot_perturbation,
		.hermite_cache = hermite_cache,
	};

	/* Zeroth order PT */
	sbmf_log_info("Starting zeroth order PT");
	f64 E0 = 0.0;
	{
		E0 += N[component] * E(&pt, component, 0);
	}
	sbmf_log_info("\tE0: %e", E0);

	/* First order PT */
	sbmf_log_info("Starting first order PT");
	f64 E1 = 0.0;
	{
		/* Handles interaction within component */
		E1 += -0.5 * G0(&pt,component,component) * N[component] * (N[component]-1) * V_closed(hermite_cache, PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), states_to_include);
	}
	sbmf_log_info("\tE1: %e", E1);

	const u32 pt2_cache_size = ((states_to_include-1)*(states_to_include))/2;
	f64 pt2_cache[pt2_cache_size];

	/* Assumes i in [0,states_to_include), j in [0,states_to_include) */
#define PT2_CACHE_INDEX(i, j) \
	((i)*states_to_include - (((i)*(i+1))/2) + j)


	/* Second order PT */
	sbmf_log_info("Starting second order PT");
	f64 E2 = 0.0;
	{
		/*
		 * Double substitutions (both excitations within same component),
		 * loop over unique pairs (j,k).
		 */
#pragma omp parallel for reduction(+: E2)
		for (u32 i = 1; i < states_to_include; ++i) {
			for (u32 j = i; j < states_to_include; ++j) {
				f64 me = rs_2nd_order_me(&pt, component,component, i,j);
				f64 Ediff = rs_2nd_order_ediff(&pt, component,component, i,j);

				pt2_cache[PT2_CACHE_INDEX(i-1, j-i)] = me/Ediff;

				E2 += me*me/(Ediff);
			}
		}
	}
	sbmf_log_info("\tE2: %e", E2);

	f64* device_pt2_cache;
	hipMalloc(&device_pt2_cache, pt2_cache_size*sizeof(f64));
	hipMemcpy(device_pt2_cache, pt2_cache, pt2_cache_size*sizeof(f64), hipMemcpyHostToDevice);

	/* Third order PT */
	sbmf_log_info("Starting third order PT");
	f64 E3 = 0.0;
	{
		f64 E_00_00 = 0;
		{
#pragma omp parallel for reduction(+: E_00_00)
			for (u32 m = 1; m < states_to_include; ++m) {
				for (u32 n = m; n < states_to_include; ++n) {
					f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1,n-m)];
					E_00_00 += tmn*tmn;
				}
			}

			f64 v_0000 = V_closed(hermite_cache, PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), states_to_include);
			E_00_00 *= G0(&pt,component,component)*v_0000;
		}
		sbmf_log_info("\t\t00,00: %.10e", E_00_00);

		f64 E_m0_n0 = 0;
		{
			const f64 c_root_2_minus_1 = sqrt(2.0) - 1.0;
			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

//#pragma omp parallel for reduction(+: E_m0_n0)
			for (u32 k = 0; k < ((states_to_include-1)*states_to_include)/2; ++k) {
				u32 m = k/(states_to_include-1);
				u32 n = k%(states_to_include-1);
				if (m > n) {
					m = (states_to_include-1) - m - 0;
					n = (states_to_include-1) - n - 1;
				}
				m += 1;
				n += 1;

				const f64 v_m0_n0 = G0(&pt,component,component)*V_closed(hermite_cache, PHI(&pt,component,m), PHI(&pt,component,0), PHI(&pt,component,n), PHI(&pt,component,0), states_to_include);

				f64 sum = 0.0;
				for (u32 p = 1; p < states_to_include; ++p) {

					f64 tmp = 0;
					if (p >= m)
						tmp = pt2_cache[PT2_CACHE_INDEX(m-1, p-m)];
					else
						tmp = pt2_cache[PT2_CACHE_INDEX(p-1, m-p)];

					f64 tnp = 0;
					if (p >= n)
						tnp = pt2_cache[PT2_CACHE_INDEX(n-1, p-n)];
					else
						tnp = pt2_cache[PT2_CACHE_INDEX(p-1, n-p)];

					const f64 delta_mp = (m == p) ? 1.0 : 0.0;
					const f64 delta_np = (n == p) ? 1.0 : 0.0;

					const f64 coeff = 1 + c_root_2_minus_1*(delta_mp + delta_np) + c_3_minus_2_root_2*(delta_mp*delta_np);
					sum += coeff * tmp * tnp;
				}

				f64 factor = (m == n) ? 1.0 : 2.0;

				E_m0_n0 += factor * v_m0_n0 * sum;
			}

			E_m0_n0 *= (N[component] - 3);
		}
		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);

		f64 E_mn_pq = 0;
		{

			const u32 num_mb_states = ((states_to_include-1)*states_to_include)/2;
			const u32 num_interactions = (num_mb_states*(num_mb_states+1))/2;

			f64* device_output;
			hipMalloc(&device_output, num_interactions*sizeof(f64));

			f64* g0;
			hipMalloc(&g0, sizeof(f64));
			hipMemcpy(g0, &pt.g0[0], sizeof(f64), hipMemcpyHostToDevice);

			const u32 blocks = (num_interactions > 256) ? num_interactions/256 : 1;
			rspt_3_mnpq<<<blocks, 256>>>(
					g0,
					states_to_include, num_mb_states, num_interactions,
					device_pt2_cache,
					hermite_cache_device,
					device_states,
					device_output
					);

			hipFree(g0);

			f64* res;
			hipMalloc(&res, sizeof(f64));
			device_sum_reduction<<<1,1>>>(res, device_output, num_interactions);
			hipMemcpy(&E_mn_pq, res, sizeof(f64), hipMemcpyDeviceToHost);
			hipFree(res);

			hipFree(device_output);
		}

		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);

		E3 = E_00_00 + E_m0_n0 + E_mn_pq;
	}
	sbmf_log_info("\tE3: %e", E3);

	hipFree(hermite_cache_device);
	hipFree(device_states);
	hipFree(device_pt2_cache);

	sbmf_stack_free_to_marker(memory_marker);

	return (struct pt_result) {
		.E0 = E0,
		.E1 = E1,
		.E2 = E2,
		.E3 = E3,
	};
}











/*********************************************************************************************/
/***								ENPT												   ***/
/*********************************************************************************************/

struct Vp_params {
	u32 coeff_count;
	f64* i;
	f64* j;
	nlse_operator_func* pert;
};

void Vp_integrand(f64* out, f64* in, u32 len, void* data);

static inline f64 en_nhn(struct pt_settings* pt, u32 A, u32 i, u32 j) {
	f64 sum = 0;
	for (u32 k = 0; k < pt->L; ++k) {
		sum += pt->states[A].eigenvectors[i*pt->L + k] * pt->states[A].eigenvectors[j*pt->L + k] * ho_eigenval(k);

	}

	/*
	 * In the case that we're dealing with a perturbation to the
	 * basis potential, we need to compute <i|Vp|j> numerically
	 * with Vp being the pertubation
	 */

	if (pt->pert) {
		struct Vp_params p = {
			.coeff_count = pt->L,
			.i = PHI(pt, A, i),
			.j = PHI(pt, A, j),
			.pert = pt->pert,
		};

		struct quadgk_settings settings = {
			.gk = gk20,
			.abs_error_tol = 1e-15,
			.rel_error_tol = 1e-15,
			.max_iters = pt->settings->max_quadgk_iters,
			.userdata = &p,
		};

		u8 quadgk_memory[quadgk_required_memory_size(&settings)];

		struct quadgk_result res;
		quadgk_infinite_interval(Vp_integrand, &settings, quadgk_memory, &res);
		assert(res.converged);

		sum += res.integral;
	}

	return sum;
}

struct pt_result enpt_1comp_cuda(struct nlse_settings settings, struct nlse_result res, u32 component, f64* g0, i64* particle_count) {
	/* order of hamiltonians, that is include all states */
	const u32 states_to_include = res.coeff_count;
	const i64* N = particle_count;
	sbmf_log_info("running 1comp ENPT:\n    components: %u\n    states: %u\n", res.component_count, states_to_include);
	const u32 A = 0;

	struct eigen_result_real states;
	states = find_eigenpairs_full_real(res.hamiltonian[component]);
	for (u32 j = 0; j < states_to_include; ++j) {
		f64_normalize(&states.eigenvectors[j*res.coeff_count], &states.eigenvectors[j*res.coeff_count], res.coeff_count);
	}

	f64* device_states;
	hipMalloc(&device_states, states_to_include*states_to_include*sizeof(f64));
	hipMemcpy(device_states, states.eigenvectors, states_to_include*states_to_include*sizeof(f64), hipMemcpyHostToDevice);
	const u64 hermite_integral_count = size4_cuda(states_to_include);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < states_to_include; ++i) {
			for (u32 j = i; j < states_to_include; ++j) {
				for (u32 k = j; k < states_to_include; ++k) {
					for (u32 l = k; l < states_to_include; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4_cuda(i,j,k,l);
					}
				}
			}
		}
	}

	struct pt_settings pt = {
		.res = &res,
		.settings = &settings,
		.g0 = g0,
		.particle_count = particle_count,
		.states = &states,
		.N = states_to_include,
		.L = res.coeff_count,
		.pert = settings.spatial_pot_perturbation,
		.hermite_cache = hermite_cache,
	};

	/* Zeroth order PT */
	sbmf_log_info("Starting zeroth order PT");
	f64 E0 = 0.0;
	{
		E0 = N[component] * en_nhn(&pt, component, 0,0) + 0.5*G0(&pt,component,component)*N[component]*(N[component]-1) *
			V_closed(hermite_cache, PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), PHI(&pt,component,0), states_to_include);
	}
	sbmf_log_info("\tE0: %e", E0);

	const u32 pt2_cache_size = ((states_to_include-1)*(states_to_include))/2;
	f64 pt2_cache[pt2_cache_size];

	/* Assumes i in [0,states_to_include), j in [0,states_to_include) */
#define PT2_CACHE_INDEX(i, j) \
	((i)*states_to_include - (((i)*(i+1))/2) + j)

	/* Second order PT */
	sbmf_log_info("Starting second order PT");
	f64 E2 = 0.0;
	{
		/*
		 * Double substitutions (both excitations within same component),
		 * loop over unique pairs (j,k).
		 */
#pragma omp parallel for reduction(+: E2)
		for (u32 m = 1; m < states_to_include; ++m) {
			for (u32 n = m; n < states_to_include; ++n) {
				f64 me = rs_2nd_order_me(&pt, component,component, m,n);
				const f64 v_mn_mn = V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,n), PHI(&pt,A,m), PHI(&pt,A,n), states_to_include);
				const f64 v_m0_m0 = V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,0), PHI(&pt,A,m), PHI(&pt,A,0), states_to_include);
				const f64 v_n0_n0 = (m == n) ? v_m0_m0 : V_closed(hermite_cache, PHI(&pt,A,n), PHI(&pt,A,0), PHI(&pt,A,n), PHI(&pt,A,0), states_to_include);
				const f64 v_00_00 = V_closed(hermite_cache, PHI(&pt,A,0), PHI(&pt,A,0), PHI(&pt,A,0), PHI(&pt,A,0), states_to_include);

				const f64 dmn = (m == n) ? 1.0 : 0.0;
				f64 Ediff =
					2.0*en_nhn(&pt,A,0,0) - en_nhn(&pt,A,m,m) - en_nhn(&pt,A,n,n)
					- G0(&pt,A,A)*((2.0-dmn)*v_mn_mn + 2.0*(N[A]-2.0)*(v_m0_m0 + v_n0_n0) - (2.0*N[A]-3.0)*v_00_00);

				pt2_cache[PT2_CACHE_INDEX(m-1, n-m)] = me/Ediff;

				E2 += me*me/(Ediff);
			}
		}
	}
	sbmf_log_info("\tE2: %e", E2);

	/* Third order PT */
	sbmf_log_info("Starting third order PT");
	f64 E3 = 0.0;
	{
		f64 E_m0_n0 = 0;
		{
			const f64 c_root_2_minus_1 = sqrt(2.0) - 1.0;
			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

#pragma omp parallel for reduction(+: E_m0_n0)
			for (u32 k = 0; k < ((states_to_include-1)*states_to_include)/2; ++k) {
				u32 m = k/(states_to_include-1);
				u32 n = k%(states_to_include-1);

				if (m > n) {
					m = (states_to_include-1) - m - 0;
					n = (states_to_include-1) - n - 1;
				}

				if (m == n)
					continue;

				m += 1;
				n += 1;

				const f64 v_m0_n0 = G0(&pt,component,component)*V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,0), PHI(&pt,A,n), PHI(&pt,A,0), states_to_include);

				f64 sum = 0.0;
				for (u32 p = 1; p < states_to_include; ++p) {

					f64 tmp = 0;
					if (p >= m)
						tmp = pt2_cache[PT2_CACHE_INDEX(m-1, p-m)];
					else
						tmp = pt2_cache[PT2_CACHE_INDEX(p-1, m-p)];

					f64 tnp = 0;
					if (p >= n)
						tnp = pt2_cache[PT2_CACHE_INDEX(n-1, p-n)];
					else
						tnp = pt2_cache[PT2_CACHE_INDEX(p-1, n-p)];

					const f64 delta_mp = (m == p) ? 1.0 : 0.0;
					const f64 delta_np = (n == p) ? 1.0 : 0.0;

					const f64 coeff = 1 + c_root_2_minus_1*(delta_mp + delta_np) + c_3_minus_2_root_2*(delta_mp*delta_np);
					sum += coeff * tmp * tnp;
				}

				E_m0_n0 += 2.0 * v_m0_n0 * sum;
			}

			E_m0_n0 *= (N[component] - 3);
		}
		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);

		f64 E_mn_pq = 0;
		{
			const f64 c_root_2_minus_2 = sqrt(2.0) - 2.0;
			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

			const u32 INDS_N = ((states_to_include-1)*states_to_include)/2;

#pragma omp parallel for reduction(+: E_mn_pq)
			for (u32 k = 0; k < INDS_N*(INDS_N+1)/2; ++k) {
				u32 k0, k1;
				map_to_triangular_index(k, INDS_N, &k0, &k1);

				/* Skip <k|V|k> terms */
				if (k0 == k1)
					continue;

				u32 m, n;
				map_to_triangular_index(k0, states_to_include-1, &m, &n);
				m += 1; n += 1;

				u32 p, q;
				map_to_triangular_index(k1, states_to_include-1, &p, &q);
				p += 1; q += 1;


				f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1,n-m)];
				const f64 delta_mn = (m == n) ? 1.0 : 0.0;

				f64 v_mn_pq = G0(&pt,component,component)*V_closed(hermite_cache, PHI(&pt,A,m), PHI(&pt,A,n), PHI(&pt,A,p), PHI(&pt,A,q), states_to_include);
				f64 tpq = pt2_cache[PT2_CACHE_INDEX(p-1,q-p)];

				const f64 delta_pq = (p == q) ? 1.0 : 0.0;
				const f64 coeff = 2.0 + c_root_2_minus_2*(delta_mn + delta_pq) + c_3_minus_2_root_2*(delta_mn*delta_pq);
				E_mn_pq += 2.0*coeff*tmn*tpq*v_mn_pq;
			}

		}
		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);

		E3 = E_m0_n0 + E_mn_pq;
	}
	sbmf_log_info("\tE3: %e", E3);

	return (struct pt_result) {
		.E0 = E0,
		.E1 = 0,
		.E2 = E2,
		.E3 = E3,
	};
}
