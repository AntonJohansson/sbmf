#include "hip/hip_runtime.h"
__host__ __device__
static inline void map_to_triangular_index(u32 k, u32 N, u32* m, u32* n) {
	*m = k / N;
	*n = k % N;
	if (*m > *n) {
		*m = N - *m - 0;
		*n = N - *n - 1;
	}
}

__host__ __device__
static inline f64 V_closed(const f64* cache, const f64* phi_a, const f64* phi_b, const f64* phi_c, const f64* phi_d, const u32 size) {
	f64 sum = 0.0;
	for (u32 a = 0; a < size; ++a) {
		for (u32 b = 0; b < size; ++b) {
			for (u32 c = 0; c < size; ++c) {
				for (u32 d = 0; d < size; ++d) {
					f64 L = phi_a[a]*phi_b[b]*phi_c[c]*phi_d[d];//*ho_K(a)*ho_K(b)*ho_K(c)*ho_K(d);
					if (fabs(L) < 1e-10)
						continue;
					f64 integral = cache[index4_cuda(a,b,c,d)];
					sum += L*integral;
				}
			}
		}
	}

	return sum;
}

/*
 * Helper functions since these will be calculated a lot
 */

__global__
static void device_sum_reduction(f64* out, f64* arr, const u32 len) {
	f64 sum = 0.0;
	for (u32 i = 0; i < len; ++i) {
		sum += arr[i];
	}
	*out = sum;
}

enum pt_mode {
	MODE_RSPT = 0,
	MODE_ENPT = 1,
};

__global__
static void rspt_3_mnpq_1comp(enum pt_mode mode, f64 g, const u32 num_sb_states, const u32 num_mb_states, const u32 num_interactions, f64* pt2_cache, f64* hermite_cache, f64* coeffs, f64* output) {
	const f64 c_root_2_minus_2 = sqrt(2.0) - 2.0;
	const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

	const u32 k = blockIdx.x*blockDim.x + threadIdx.x;
	if (k >= num_interactions)
		return;
	u32 k0, k1;
	map_to_triangular_index(k, num_mb_states, &k0, &k1);

	f64 factor = 2.0;
	if (k0 == k1) {
		if (mode == MODE_RSPT)
			factor = 1.0;
		else if (mode == MODE_ENPT)
			return;
	}

	u32 m, n;
	map_to_triangular_index(k0, num_sb_states-1, &m, &n);
	m += 1; n += 1;

	u32 p, q;
	map_to_triangular_index(k1, num_sb_states-1, &p, &q);
	p += 1; q += 1;

	const f64 tmn = pt2_cache[index2_cuda(m-1,n-1)];
	const f64 tpq = pt2_cache[index2_cuda(p-1,q-1)];

	const f64 delta_mn = (m == n) ? 1.0 : 0.0;
	const f64 delta_pq = (p == q) ? 1.0 : 0.0;

	f64 v_mn_pq = g*V_closed(hermite_cache,
			&coeffs[m*num_sb_states],
			&coeffs[n*num_sb_states],
			&coeffs[p*num_sb_states],
			&coeffs[q*num_sb_states],
			num_sb_states);


	const f64 coeff = 2.0 + c_root_2_minus_2*(delta_mn + delta_pq) + c_3_minus_2_root_2*(delta_mn*delta_pq);
	output[k] = factor*coeff*tmn*tpq*v_mn_pq;
}

__global__
static void rspt_3_mnpq_2comp(enum pt_mode mode, f64 g, const u32 num_sb_states, const u32 num_mb_states, const u32 num_interactions, f64* pt2_cache, f64* hermite_cache, f64* coeffsA, f64* coeffsB, f64* output) {
	const u32 k = blockIdx.x*blockDim.x + threadIdx.x;
	if (k >= num_interactions)
		return;

	u32 k0, k1;
	map_to_triangular_index(k, num_mb_states, &k0, &k1);

	f64 factor = 2.0;
	if (k0 == k1) {
		if (mode == MODE_RSPT)
			factor = 1.0;
		else if (mode == MODE_ENPT) {
			output[k] = 0;
			return;
		}
	}

	const u32 m = k0 % (num_sb_states-1) + 1;
	const u32 n = k0 / (num_sb_states-1) + 1;
	const u32 p = k1 % (num_sb_states-1) + 1;
	const u32 q = k1 / (num_sb_states-1) + 1;

#define PT2_CACHE_INDEX(i, j) \
	(i)*(num_sb_states-1) + (j)

	const f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1, n-1)];
	const f64 tpq = pt2_cache[PT2_CACHE_INDEX(p-1, q-1)];
#undef PT2_CACHE_INDEX

	const f64 v_mn_pq = g * V_closed(hermite_cache,
			&coeffsA[m*num_sb_states],
			&coeffsB[n*num_sb_states],
			&coeffsA[p*num_sb_states],
			&coeffsB[q*num_sb_states],
			num_sb_states);
	output[k] = factor*v_mn_pq*tmn*tpq;
}

struct Vp_params {
	u32 coeff_count;
	f64* i;
	f64* j;
	nlse_operator_func* pert;
};

void Vp_integrand(f64* out, f64* in, u32 len, void* data);

static inline f64 en_nhn_new(f64* phi_m, f64* phi_n, const u32 num_sb_states, nlse_operator_func* pert) {
	f64 sum = 0;
	for (u32 k = 0; k < num_sb_states; ++k) {
		sum += phi_m[k]*phi_n[k]*ho_eigenval(k);

	}

	/*
	 * In the case that we're dealing with a perturbation to the
	 * basis potential, we need to compute <i|Vp|j> numerically
	 * with Vp being the pertubation
	 */

	if (pert) {
		struct Vp_params p = {
			.coeff_count = num_sb_states,
			.i = phi_m,
			.j = phi_n,
			.pert = pert,
		};

		struct quadgk_settings settings = {
			.gk = gk20,
			.abs_error_tol = 1e-15,
			.rel_error_tol = 1e-15,
			.max_iters = 500,
			.userdata = &p,
		};

		u8 quadgk_memory[quadgk_required_memory_size(&settings)];

		struct quadgk_result res;
		quadgk_infinite_interval(Vp_integrand, &settings, quadgk_memory, &res);
		assert(res.converged);

		sum += res.integral;
	}

	return sum;
}



/**************************************************************************************************************************************************/




static struct pt_result perturbation_theory_1comp(enum pt_mode mode, f64 g, i64 N, const f64* hermite_cache, const u32 hermite_cache_size, const struct eigen_result_real* states, const f64 groundstate_energy, const f64* double_subst_energy_diffs, const u32 num_sb_states) {
	f64* device_states;
	hipMalloc(&device_states, num_sb_states*num_sb_states*sizeof(f64));
	hipMemcpy(device_states, states->eigenvectors, num_sb_states*num_sb_states*sizeof(f64), hipMemcpyHostToDevice);

	f64* hermite_cache_device;
	hipMalloc(&hermite_cache_device, hermite_cache_size);
	hipMemcpy(hermite_cache_device, hermite_cache, hermite_cache_size, hipMemcpyHostToDevice);

	/* Zeroth order PT */
	sbmf_log_info("Starting zeroth order PT");
	f64 E0 = groundstate_energy;
	sbmf_log_info("\tE0: %e", E0);

	/* This particular integral shows up in zeroth and third order rspt */
	const f64 v_00_00 = V_closed(hermite_cache,
			&states->eigenvectors[0],
			&states->eigenvectors[0],
			&states->eigenvectors[0],
			&states->eigenvectors[0],
			num_sb_states);

	/* First order PT */
	f64 E1 = 0.0;
	if (mode == MODE_RSPT) {
		sbmf_log_info("Starting first order PT");
		E1 = -0.5*g*N*(N-1)*v_00_00;
		sbmf_log_info("\tE1: %e", E1);
	}

	const u32 pt2_cache_size = size2_cuda(num_sb_states-1);
	f64 pt2_cache[pt2_cache_size];

	/* Second order PT */
	sbmf_log_info("Starting second order PT");
	f64 E2 = 0.0;
	{
		/*
		 * Double substitutions (both excitations within same component),
		 * loop over unique pairs (j,k).
		 */
#pragma omp parallel for reduction(+: E2)
		for (u32 m = 1; m < num_sb_states; ++m) {
			for (u32 n = m; n < num_sb_states; ++n) {
				const f64 v_mn_00 = V_closed(hermite_cache,
						&states->eigenvectors[m*num_sb_states],
						&states->eigenvectors[n*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						num_sb_states);
				const f64 factor = (m == n) ? 1.0/sqrt(2.0) : 1.0;
				const f64 me = factor*g*sqrt(N*(N-1))*v_mn_00;

				const f64 Ediff = double_subst_energy_diffs[index2_cuda(m-1,n-1)];

				pt2_cache[index2_cuda(m-1,n-1)] = me/Ediff;

				E2 += me*me/(Ediff);
			}
		}
	}
	sbmf_log_info("\tE2: %e", E2);

	f64* device_pt2_cache;
	hipMalloc(&device_pt2_cache, pt2_cache_size*sizeof(f64));
	hipMemcpy(device_pt2_cache, pt2_cache, pt2_cache_size*sizeof(f64), hipMemcpyHostToDevice);

	/* Third order PT */
	sbmf_log_info("Starting third order PT");
	f64 E3 = 0.0;
	{
		f64 E_00_00 = 0;
		if (mode == MODE_RSPT) {
#pragma omp parallel for reduction(+: E_00_00)
			for (u32 m = 1; m < num_sb_states; ++m) {
				for (u32 n = m; n < num_sb_states; ++n) {
					const f64 tmn = pt2_cache[index2_cuda(m-1,n-1)];
					E_00_00 += tmn*tmn;
				}
			}

			E_00_00 *= g*v_00_00;
			sbmf_log_info("\t\t00,00: %.10e", E_00_00);
		}

		/* Number of many-body states, excludes 0,0 */
		const u32 num_mb_states = size2_cuda(num_sb_states-1);

		f64 E_m0_n0 = 0;
		{
			const f64 c_root_2_minus_1 = sqrt(2.0) - 1.0;
			const f64 c_3_minus_2_root_2 = 3.0 - 2.0*sqrt(2.0);

#pragma omp parallel for reduction(+: E_m0_n0)
			for (u32 k = 0; k < num_mb_states; ++k) {
				u32 m, n;
				map_to_triangular_index(k, num_sb_states-1, &m, &n);
				if (mode == MODE_ENPT && m == n)
					continue;

				m += 1;
				n += 1;

				const f64 v_m0_n0 = g*V_closed(hermite_cache,
						&states->eigenvectors[m*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						&states->eigenvectors[n*num_sb_states],
						&states->eigenvectors[0*num_sb_states],
						num_sb_states);

				f64 sum = 0.0;
				for (u32 p = 1; p < num_sb_states; ++p) {

					const f64 tmp = pt2_cache[index2_cuda(m-1,p-1)];
					const f64 tnp = pt2_cache[index2_cuda(n-1,p-1)];

					const f64 delta_mp = (m == p) ? 1.0 : 0.0;
					const f64 delta_np = (n == p) ? 1.0 : 0.0;

					const f64 coeff = 1 + c_root_2_minus_1*(delta_mp + delta_np) + c_3_minus_2_root_2*(delta_mp*delta_np);
					sum += coeff * tmp * tnp;
				}

				f64 factor = (m == n) ? 1.0 : 2.0;

				E_m0_n0 += factor * v_m0_n0 * sum;
			}

			E_m0_n0 *= (N - 3);
		}
		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);

		f64 E_mn_pq = 0;
		{
			const u32 num_interactions = (num_mb_states*(num_mb_states+1))/2;

			f64* device_output;
			hipMalloc(&device_output, num_interactions*sizeof(f64));

			const u32 blocks = num_interactions/256 + 1;
			rspt_3_mnpq_1comp<<<blocks, 256>>>(
					mode,
					g,
					num_sb_states, num_mb_states, num_interactions,
					device_pt2_cache,
					hermite_cache_device,
					device_states,
					device_output
					);

			f64* res;
			hipMalloc(&res, sizeof(f64));
			device_sum_reduction<<<1,1>>>(res, device_output, num_interactions);
			hipMemcpy(&E_mn_pq, res, sizeof(f64), hipMemcpyDeviceToHost);
			hipFree(res);

			hipFree(device_output);
		}

		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);

		E3 = E_00_00 + E_m0_n0 + E_mn_pq;
	}
	sbmf_log_info("\tE3: %e", E3);

	hipFree(hermite_cache_device);
	hipFree(device_states);
	hipFree(device_pt2_cache);

	return (struct pt_result) {
		.E0 = E0,
		.E1 = E1,
		.E2 = E2,
		.E3 = E3,
	};
}

static struct pt_result perturbation_theory_2comp(enum pt_mode mode, f64 gAA, f64 gAB, i64 NA, i64 NB, const f64* hermite_cache, const u32 hermite_cache_size, const struct eigen_result_real* statesA, const struct eigen_result_real* statesB,
		const f64 groundstate_energy,
		const f64* double_subst_energy_diffs_AA,
		const f64* double_subst_energy_diffs_BB,
		const f64* double_subst_energy_diffs_AB,
		const u32 num_sb_states) {

	f64* device_states_A;
	f64* device_states_B;
	hipMalloc(&device_states_A, num_sb_states*num_sb_states*sizeof(f64));
	hipMemcpy(device_states_A, statesA->eigenvectors, num_sb_states*num_sb_states*sizeof(f64), hipMemcpyHostToDevice);
	hipMalloc(&device_states_B, num_sb_states*num_sb_states*sizeof(f64));
	hipMemcpy(device_states_B, statesB->eigenvectors, num_sb_states*num_sb_states*sizeof(f64), hipMemcpyHostToDevice);

	f64* hermite_cache_device;
	hipMalloc(&hermite_cache_device, hermite_cache_size);
	hipMemcpy(hermite_cache_device, hermite_cache, hermite_cache_size, hipMemcpyHostToDevice);

	struct pt_result res_A = perturbation_theory_1comp(mode, gAA, NA, hermite_cache, hermite_cache_size, statesA, groundstate_energy, double_subst_energy_diffs_AA, num_sb_states);
	struct pt_result res_B = perturbation_theory_1comp(mode, gAA, NB, hermite_cache, hermite_cache_size, statesB, groundstate_energy, double_subst_energy_diffs_BB, num_sb_states);

	sbmf_log_info("Starting zeroth order PT");
	//f64 E0 = res_A.E0 + res_B.E0;
	f64 E0 = groundstate_energy;
	sbmf_log_info("\tE0: %e", E0);

	f64 E1 = 0.0;
	if (mode == MODE_RSPT) {
		sbmf_log_info("Starting first order PT");
		E1 = res_A.E1 + res_B.E1;
		E1 += -gAB*NA*NB*V_closed(hermite_cache,
				&statesA->eigenvectors[0],
				&statesB->eigenvectors[0],
				&statesA->eigenvectors[0],
				&statesB->eigenvectors[0],
				num_sb_states);
		sbmf_log_info("\tE1: %e", E1);
	}

	const u32 pt2_cache_size = (num_sb_states-1)*(num_sb_states-1);
	f64 pt2_cache[pt2_cache_size];

	/* Assumes i in [0,states_to_include), j in [0,states_to_include) */
#define PT2_CACHE_INDEX(i, j) \
	(i)*(num_sb_states-1) + (j)

	/* Second order PT */
	sbmf_log_info("Starting second order PT");
	f64 E2 = 0.0;
	{
		E2 = res_A.E2 + res_B.E2;
		/*
		 * Double substitutions (separate components).
		 * A,B refers to components.
		 */
#pragma omp parallel for reduction(+: E2)
		for (u32 m = 1; m < num_sb_states; ++m) {
			for (u32 n = 1; n < num_sb_states; ++n) {
				const f64 me = gAB*sqrt(NA*NB)
					* V_closed(hermite_cache,
							&statesA->eigenvectors[m*num_sb_states],
							&statesB->eigenvectors[n*num_sb_states],
							&statesA->eigenvectors[0*num_sb_states],
							&statesB->eigenvectors[0*num_sb_states],
							num_sb_states);

				const f64 Ediff = double_subst_energy_diffs_AB[(m-1)*(num_sb_states-1) + (n-1)];

				pt2_cache[PT2_CACHE_INDEX(m-1,n-1)] = me/Ediff;

				E2 += me*me/(Ediff);
			}
		}
	}
	sbmf_log_info("\tE2: %e", E2);

	f64* device_pt2_cache;
	hipMalloc(&device_pt2_cache, pt2_cache_size*sizeof(f64));
	hipMemcpy(device_pt2_cache, pt2_cache, pt2_cache_size*sizeof(f64), hipMemcpyHostToDevice);

	sbmf_log_info("Starting third order PT");
	f64 E3 = 0.0;
	{
		E3 = res_A.E3 + res_B.E3;

		f64 E_00_00 = 0.0;
		if (mode == MODE_RSPT) {

			f64 sum = 0;
#pragma omp parallel for reduction(+: sum)
			for (u32 m = 1; m < num_sb_states; ++m) {
				for (u32 n = 1; n < num_sb_states; ++n) {
					const f64 tmn = pt2_cache[PT2_CACHE_INDEX(m-1, n-1)];
					sum += tmn*tmn;
				}
			}

			const f64 v_00_00 = V_closed(hermite_cache,
					&statesA->eigenvectors[0],
					&statesB->eigenvectors[0],
					&statesA->eigenvectors[0],
					&statesB->eigenvectors[0],
					num_sb_states);
			E_00_00 += gAB * v_00_00 * sum;
		}
		sbmf_log_info("\t\t00,00: %.10e", E_00_00);

		f64 E_m0_n0 = 0.0;
		{
			{
#pragma omp parallel for collapse(2) reduction(+: E_m0_n0)
				for (u32 m = 1; m < num_sb_states; ++m) {
					for (u32 n = 1; n < num_sb_states; ++n) {
						if (mode == MODE_ENPT && m == n)
							continue;

						f64 sumA = 0;
						f64 sumB = 0;
						for (u32 p = 1; p < num_sb_states; ++p) {
							const f64 tmp = pt2_cache[PT2_CACHE_INDEX(m-1, p-1)];
							const f64 tnp = pt2_cache[PT2_CACHE_INDEX(n-1, p-1)];
							sumA += tmp*tnp;

							const f64 tpm = pt2_cache[PT2_CACHE_INDEX(p-1, m-1)];
							const f64 tpn = pt2_cache[PT2_CACHE_INDEX(p-1, n-1)];
							sumB += tpm*tpn;
						}

						const f64 v_AA_m0_n0 = gAA*(NA-1)*V_closed(hermite_cache,
								&statesA->eigenvectors[m*num_sb_states],
								&statesA->eigenvectors[0*num_sb_states],
								&statesA->eigenvectors[n*num_sb_states],
								&statesA->eigenvectors[0*num_sb_states],
								num_sb_states);
						const f64 v_BB_m0_n0 = gAA*(NB-1)*V_closed(hermite_cache,
								&statesB->eigenvectors[m*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								&statesB->eigenvectors[n*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								num_sb_states);
						const f64 v_AB_m0_n0 = - gAB*V_closed(hermite_cache,
								&statesA->eigenvectors[m*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								&statesA->eigenvectors[n*num_sb_states],
								&statesB->eigenvectors[0*num_sb_states],
								num_sb_states);
						const f64 v_AB_0m_0n = - gAB*V_closed(hermite_cache,
								&statesA->eigenvectors[0*num_sb_states],
								&statesB->eigenvectors[m*num_sb_states],
								&statesA->eigenvectors[0*num_sb_states],
								&statesB->eigenvectors[n*num_sb_states],
								num_sb_states);
						E_m0_n0 += (v_AA_m0_n0 + v_AB_m0_n0) * sumA;
						E_m0_n0 += (v_BB_m0_n0 + v_AB_0m_0n) * sumB;
					}
				}
			}
		}
		sbmf_log_info("\t\tm0,n0: %.10e", E_m0_n0);

		f64 E_mn_pq = 0;
		{
			const u32 num_mb_states = (num_sb_states-1)*(num_sb_states-1);
			const u32 num_interactions = (num_mb_states*(num_mb_states+1))/2;

			f64* device_output;
			hipMalloc(&device_output, num_interactions*sizeof(f64));

			const u32 blocks = num_interactions/256 + 1;
			rspt_3_mnpq_2comp<<<blocks, 256>>>(
					mode,
					gAB,
					num_sb_states, num_mb_states, num_interactions,
					device_pt2_cache,
					hermite_cache_device,
					device_states_A,
					device_states_B,
					device_output
					);

			f64* res;
			hipMalloc(&res, sizeof(f64));
			device_sum_reduction<<<1,1>>>(res, device_output, num_interactions);
			hipMemcpy(&E_mn_pq, res, sizeof(f64), hipMemcpyDeviceToHost);
			hipFree(res);

			hipFree(device_output);
		}
		sbmf_log_info("\t\tmn,pq: %.10e", E_mn_pq);

		E3 += E_00_00 + E_m0_n0 + E_mn_pq;
	}
	sbmf_log_info("\tE3: %e", E3);

	hipFree(device_pt2_cache);
	hipFree(hermite_cache_device);
	hipFree(device_states_A);
	hipFree(device_states_B);

	return (struct pt_result) {
		.E0 = E0,
		.E1 = E1,
		.E2 = E2,
		.E3 = E3,
	};
}

/**************************************************************************************************************************************************/

struct pt_result rspt_1comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 component, f64 g, i64 N) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real states;
	states = find_eigenpairs_full_real(res.hamiltonian[component]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&states.eigenvectors[j*num_sb_states], &states.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4(i,j,k,l);
					}
				}
			}
		}
	}


	f64 groundstate_energy = N*states.eigenvalues[0];
	/* Energies of double substitution states including the zero states */
	f64 double_subst_energy_diffs[size2_cuda(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			double_subst_energy_diffs[index2_cuda(m-1,n-1)] = 2*states.eigenvalues[0] - states.eigenvalues[m] - states.eigenvalues[n];
		}
	}

	struct pt_result ptres = perturbation_theory_1comp(MODE_RSPT, g, N, hermite_cache, hermite_cache_size, &states, groundstate_energy, double_subst_energy_diffs, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}

struct pt_result enpt_1comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 component, f64 g, i64 N) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real states;
	states = find_eigenpairs_full_real(res.hamiltonian[component]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&states.eigenvectors[j*num_sb_states], &states.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4(i,j,k,l);
					}
				}
			}
		}
	}

	/* Energies of double substitution states including the zero states */
	f64 double_subst_energy_diffs[size2_cuda(num_sb_states-1)];

	const f64 v_00_00 = V_closed(hermite_cache,
			&states.eigenvectors[0],
			&states.eigenvectors[0],
			&states.eigenvectors[0],
			&states.eigenvectors[0],
			num_sb_states);
	f64 groundstate_energy = N*en_nhn_new(&states.eigenvectors[0*num_sb_states], &states.eigenvectors[0*num_sb_states], num_sb_states, settings->spatial_pot_perturbation) + 0.5*g*N*(N-1)*v_00_00;

	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			const f64 v_mn_mn = V_closed(hermite_cache,
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[n*num_sb_states],
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_m0_m0 = V_closed(hermite_cache,
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					&states.eigenvectors[m*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_n0_n0 = (m == n) ? v_m0_m0 : V_closed(hermite_cache,
					&states.eigenvectors[n*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					&states.eigenvectors[n*num_sb_states],
					&states.eigenvectors[0*num_sb_states],
					num_sb_states);

			const f64 dmn = (m == n) ? 1.0 : 0.0;
			f64 energy =
				2*en_nhn_new(&states.eigenvectors[0*num_sb_states], &states.eigenvectors[0*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				-en_nhn_new(&states.eigenvectors[m*num_sb_states], &states.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				-en_nhn_new(&states.eigenvectors[n*num_sb_states], &states.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				- g*((2.0-dmn)*v_mn_mn + 2.0*(N-2)*(v_m0_m0+v_n0_n0) - (2*N-3)*v_00_00);

			double_subst_energy_diffs[index2_cuda(m-1,n-1)] = energy;
		}
	}

	struct pt_result ptres = perturbation_theory_1comp(MODE_ENPT, g, N, hermite_cache, hermite_cache_size, &states, groundstate_energy, double_subst_energy_diffs, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}

struct pt_result rspt_2comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 compA, u32 compB, f64 gAA, f64 gAB, i64 NA, i64 NB) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real statesA, statesB;
	statesA = find_eigenpairs_full_real(res.hamiltonian[compA]);
	statesB = find_eigenpairs_full_real(res.hamiltonian[compB]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&statesA.eigenvectors[j*num_sb_states], &statesA.eigenvectors[j*num_sb_states], num_sb_states);
		f64_normalize(&statesB.eigenvectors[j*num_sb_states], &statesB.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4(i,j,k,l);
					}
				}
			}
		}
	}

	const f64 groundstate_energy = NA*statesA.eigenvalues[0] + NB*statesB.eigenvalues[0];

	/* Energies of double substitution states including the zero states */
	f64 double_subst_energies_AA[size2_cuda(num_sb_states-1)];
	f64 double_subst_energies_BB[size2_cuda(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			double_subst_energies_AA[index2_cuda(m-1,n-1)] = 2*statesA.eigenvalues[0] - statesA.eigenvalues[m] - statesA.eigenvalues[n];
			double_subst_energies_BB[index2_cuda(m-1,n-1)] = 2*statesB.eigenvalues[0] - statesB.eigenvalues[m] - statesB.eigenvalues[n];
		}
	}

	f64 double_subst_energies_AB[(num_sb_states-1)*(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = 1; n < num_sb_states; ++n) {
			double_subst_energies_AB[(m-1)*(num_sb_states-1) + (n-1)] = statesA.eigenvalues[0] + statesB.eigenvalues[0] - statesA.eigenvalues[m] - statesB.eigenvalues[n];
		}
	}

	struct pt_result ptres = perturbation_theory_2comp(MODE_RSPT, gAA, gAB, NA, NB, hermite_cache, hermite_cache_size, &statesA, &statesB, groundstate_energy, double_subst_energies_AA, double_subst_energies_BB, double_subst_energies_AB, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}

struct pt_result enpt_2comp_cuda_new(struct nlse_settings* settings, struct nlse_result res, u32 compA, u32 compB, f64 gAA, f64 gAB, i64 NA, i64 NB) {
	/*
	 * The number of single body (sb) states is equal to the number
	 * of coefficients which is equal to the basis size
	 */
	const u32 num_sb_states = res.coeff_count;

	/* Find all eigenstates and eigenenergies of the hamiltonian passed in */
	struct eigen_result_real statesA, statesB;
	statesA = find_eigenpairs_full_real(res.hamiltonian[compA]);
	statesB = find_eigenpairs_full_real(res.hamiltonian[compB]);
	for (u32 j = 0; j < num_sb_states; ++j) {
		f64_normalize(&statesA.eigenvectors[j*num_sb_states], &statesA.eigenvectors[j*num_sb_states], num_sb_states);
		f64_normalize(&statesB.eigenvectors[j*num_sb_states], &statesB.eigenvectors[j*num_sb_states], num_sb_states);
	}

	const u64 hermite_integral_count = size4_cuda(num_sb_states);
	const u64 hermite_cache_size = sizeof(f64)*hermite_integral_count;
	u32 memory_marker = sbmf_stack_marker();
	f64* hermite_cache = (f64*)sbmf_stack_push(hermite_cache_size);
	{
		sbmf_log_info("Precomputing %ld hermite integrals", hermite_integral_count);
		for (u32 i = 0; i < num_sb_states; ++i) {
			for (u32 j = i; j < num_sb_states; ++j) {
				for (u32 k = j; k < num_sb_states; ++k) {
					for (u32 l = k; l < num_sb_states; ++l) {
						hermite_cache[index4_cuda(i,j,k,l)] = hermite_integral_4(i,j,k,l);
					}
				}
			}
		}
	}

	const f64 v_AA_00_00 = V_closed(hermite_cache,
			&statesA.eigenvectors[0],
			&statesA.eigenvectors[0],
			&statesA.eigenvectors[0],
			&statesA.eigenvectors[0],
			num_sb_states);
	const f64 v_BB_00_00 = V_closed(hermite_cache,
			&statesB.eigenvectors[0],
			&statesB.eigenvectors[0],
			&statesB.eigenvectors[0],
			&statesB.eigenvectors[0],
			num_sb_states);
	const f64 v_AB_00_00 = V_closed(hermite_cache,
			&statesA.eigenvectors[0],
			&statesB.eigenvectors[0],
			&statesA.eigenvectors[0],
			&statesB.eigenvectors[0],
			num_sb_states);

	const f64 groundstate_energy =
		  NA*en_nhn_new(&statesA.eigenvectors[0], &statesA.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation) + 0.5*gAA*NA*(NA-1)*v_AA_00_00
		+ NB*en_nhn_new(&statesB.eigenvectors[0], &statesB.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation) + 0.5*gAA*NB*(NB-1)*v_BB_00_00
		+ gAB*NA*NB*v_AB_00_00;

	/* Energies of double substitution states including the zero states */
	f64 double_subst_energy_diffs_AA[size2_cuda(num_sb_states-1)];
	f64 double_subst_energy_diffs_BB[size2_cuda(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = m; n < num_sb_states; ++n) {
			f64 delta_mn = (m == n) ? 1.0 : 0.0;
			const f64 v_AA_mn_mn = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_AA_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AA_n0_n0 = (m == n) ? v_AA_m0_m0 : V_closed(hermite_cache,
					&statesA.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_n0_n0 = (m == n) ? v_AB_m0_m0 : V_closed(hermite_cache,
					&statesA.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			double_subst_energy_diffs_AA[index2_cuda(m-1,n-1)] =
				  2*en_nhn_new(&statesA.eigenvectors[0], &statesA.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesA.eigenvectors[m*num_sb_states], &statesA.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesA.eigenvectors[n*num_sb_states], &statesA.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*((2.0-delta_mn)*v_AA_mn_mn + 2.0*(NA-2)*(v_AA_m0_m0 + v_AA_n0_n0) - (2.0*NA-3.0)*v_AA_00_00)
				  - gAB*NB*(v_AB_m0_m0 + v_AB_n0_n0 - 2.0*v_AB_00_00);


			const f64 v_BB_mn_mn = V_closed(hermite_cache,
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_BB_m0_m0 = V_closed(hermite_cache,
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_BB_n0_n0 = (m == n) ? v_AA_m0_m0 : V_closed(hermite_cache,
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_0m_0m = V_closed(hermite_cache,
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[m*num_sb_states],
					num_sb_states);
			const f64 v_AB_0n_0n = (m == n) ? v_AB_m0_m0 : V_closed(hermite_cache,
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			double_subst_energy_diffs_BB[index2_cuda(m-1,n-1)] =
				  2*en_nhn_new(&statesB.eigenvectors[0], &statesB.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesB.eigenvectors[m*num_sb_states], &statesB.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesB.eigenvectors[n*num_sb_states], &statesB.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*((2.0-delta_mn)*v_BB_mn_mn + 2.0*(NB-2)*(v_BB_m0_m0 + v_BB_n0_n0) - (2.0*NB-3.0)*v_BB_00_00)
				  - gAB*NB*(v_AB_0m_0m + v_AB_0n_0n - 2.0*v_AB_00_00);
		}
	}

	f64 double_subst_energy_diffs_AB[(num_sb_states-1)*(num_sb_states-1)];
	for (u32 m = 1; m < num_sb_states; ++m) {
		for (u32 n = 1; n < num_sb_states; ++n) {
			const f64 v_AA_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_BB_n0_n0 = V_closed(hermite_cache,
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_mn_mn = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			const f64 v_AB_m0_m0 = V_closed(hermite_cache,
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					&statesA.eigenvectors[m*num_sb_states],
					&statesB.eigenvectors[0*num_sb_states],
					num_sb_states);
			const f64 v_AB_0n_0n = V_closed(hermite_cache,
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					&statesA.eigenvectors[0*num_sb_states],
					&statesB.eigenvectors[n*num_sb_states],
					num_sb_states);
			double_subst_energy_diffs_AB[(m-1)*(num_sb_states-1) + (n-1)] =
				    en_nhn_new(&statesA.eigenvectors[0], &statesA.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesA.eigenvectors[m*num_sb_states], &statesA.eigenvectors[m*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*(2.0*(NA-1)*v_AA_m0_m0 - (NA-1)*v_AA_00_00)
				  + en_nhn_new(&statesB.eigenvectors[0], &statesB.eigenvectors[0], num_sb_states, settings->spatial_pot_perturbation)
				  - en_nhn_new(&statesB.eigenvectors[n*num_sb_states], &statesB.eigenvectors[n*num_sb_states], num_sb_states, settings->spatial_pot_perturbation)
				  - gAA*(2.0*(NB-1)*v_BB_n0_n0 - (NB-1)*v_BB_00_00)
				  - gAB*(v_AB_mn_mn + (NB-1)*v_AB_m0_m0 + (NA-1)*v_AB_0n_0n - (NA + NB - 1)*v_AB_00_00);
		}
	}

	struct pt_result ptres = perturbation_theory_2comp(MODE_ENPT, gAA, gAB, NA, NB, hermite_cache, hermite_cache_size, &statesA, &statesB, groundstate_energy, double_subst_energy_diffs_AA, double_subst_energy_diffs_BB, double_subst_energy_diffs_AB, num_sb_states);
	sbmf_stack_free_to_marker(memory_marker);

	return ptres;
}
